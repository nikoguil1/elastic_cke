#include "hip/hip_runtime.h"
#include <unistd.h>
#include <time.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>   // helper functions for string parsing
#include <hip/hip_runtime_api.h>   
#include "elastic_kernel.h"

int main(int argc, char **argv)
{
	t_kernel_stub *kstub;
	
	if (argc <4) {
		printf("Error: program must run as follows: sheduler device_id kernel_name num_BpSM iterations\n");
		return -1;
	}
	
	int deviceId = atoi(argv[1]);
	
	hipSetDevice(deviceId);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId); 
	
	/** Create commom streams for all kernels: two for asynchronous transfers, one for preemption commands*/
	hipStream_t *transfers_s;
	transfers_s = (hipStream_t *)calloc(2, sizeof(hipStream_t));
	
	for (int i=0;i<2;i++){
		hipStreamCreate(&transfers_s[i]);
	} 
	
	hipStream_t preemp_s;
	checkCudaErrors(hipStreamCreateWithFlags(&preemp_s, hipStreamNonBlocking)); 
	
	//cudaProfilerInitialize("profiling_options.txt", "sal_prof", cudaKeyValuePair );

	t_Kernel kid=(t_Kernel)-1;
	
	int flag_create_kstub_with_params = 0;
	
	if (strcmp(argv[2], "MM") == 0){
		kid = MM;
	}
	
	if (strcmp(argv[2], "BS") == 0){
		kid = BS;
	}
		
	if (strcmp(argv[2], "VA") == 0){
		kid = VA;
	}
	
	if (strcmp(argv[2], "PF") == 0){
		kid = PF;
	}
	
	if (strcmp(argv[2], "SPMV_CSRscalar") == 0){
		kid = SPMV_CSRscalar;
	}
	
	if (strcmp(argv[2], "PF") == 0){
		kid = PF;
	}
	
	if (strcmp(argv[2], "RCONV") == 0){
		kid = RCONV;
	}
	
	if (strcmp(argv[2], "CCONV") == 0){
		
		// First RCONV must be executed 
		kid = RCONV;
		
		/** Create stub ***/
		create_stubinfo(&kstub, deviceId, kid, transfers_s, &preemp_s);
	
		// make HtD transfers 
		(kstub->startMallocs)((void *)(kstub));
		(kstub->startTransfers)((void *)(kstub));	
		hipDeviceSynchronize();
		
		// Exec 
		int idSMs[2];
		
		idSMs[0]=0;idSMs[1]=kstub->kconf.numSMs-1;
		kstub->idSMs = idSMs;
		(kstub->launchCKEkernel)(kstub);
		hipDeviceSynchronize();
		
		flag_create_kstub_with_params = 1;
		kid = CCONV;
	}
	
	if (strcmp(argv[2], "GCEDD") == 0){
		kid = GCEDD;
	}
	
	if (strcmp(argv[2], "SCEDD") == 0){
		
		// First GCEDD must be executed 
		kid = GCEDD;
		/** Create stub ***/
		create_stubinfo(&kstub, deviceId, kid, transfers_s, &preemp_s);
	
		// make HtD transfers 
		(kstub->startMallocs)((void *)(kstub));
		(kstub->startTransfers)((void *)(kstub));	
		hipDeviceSynchronize();
		
		// Exec 
		int idSMs[2];
		
		idSMs[0]=0;idSMs[1]=kstub->kconf.numSMs-1;
		kstub->idSMs = idSMs;
		(kstub->launchCKEkernel)(kstub);
		hipDeviceSynchronize();
		
		flag_create_kstub_with_params = 1;
		kid = SCEDD;

	}
	
	if (strcmp(argv[2], "Reduction") == 0){
		kid = Reduction;
	}
	
	if (strcmp(argv[2], "HST256") == 0){
		kid = HST256;
	}

	if (kid < 0){
		printf("Error: Wrong kernel name\n");
		return -1;
	}
	
	/** Create stub ***/
	t_kernel_stub *kstub1;
	
	if (flag_create_kstub_with_params == 0)
		create_stubinfo(&kstub1, deviceId, kid, transfers_s, &preemp_s);
	else
		create_stubinfo_with_params(&kstub1, deviceId, kid, transfers_s, &preemp_s, (void *)kstub->params);
	
	// make HtD transfers 
	(kstub1->startMallocs)((void *)(kstub1));
	(kstub1->startTransfers)((void *)(kstub1));
	
	hipDeviceSynchronize();
	
	// Solo execution
	
	int BpSM = atoi(argv[3]);
	int iterations = atoi(argv[4]);
	struct timespec now;
	int idSMs[2];
	double time1, time2;
		
	idSMs[0]=0;idSMs[1]=kstub1->kconf.numSMs-1;
	kstub1->idSMs = idSMs;	
	
	kstub1->kconf.max_persistent_blocks = BpSM; // Limit the max number of blocks per SM
	
	for (int i=0; i<iterations; i++) {
	
		clock_gettime(CLOCK_REALTIME, &now);
		time1 = (double)now.tv_sec+(double)now.tv_nsec*1e-9;
		
		hipProfilerStart();
	
		(kstub1->launchCKEkernel)((void *)kstub1);
		hipDeviceSynchronize();
		
		hipProfilerStop();

		clock_gettime(CLOCK_REALTIME, &now);
		time2 = (double)now.tv_sec+(double)now.tv_nsec*1e-9;
	
		int exec_tasks=0;
		hipMemcpyAsync(kstub1->d_executed_tasks, &exec_tasks, sizeof(int), hipMemcpyHostToDevice, *kstub1->preemp_s); // Reset task counter
		
		hipDeviceSynchronize();
	}
			
	printf("BSP=%d Time=%f Tpms=%f\n", BpSM, time2-time1, (double)kstub1->total_tasks/(1000.0*(time2-time1)));
	
	
	return 0;
}