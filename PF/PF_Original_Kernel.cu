#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>

// Files
#include <sstream>
#include <string>
#include <fstream>
#include <iostream>

// Vectors
#include <vector>

using namespace std;

#include "../elastic_kernel.h"
#include "PF.h"

#define HALO 1 // halo width along one direction when advancing to the next iteration

int rows, cols;
int* data;
int** wall;
int* result;
#define M_SEED 9
int pyramid_height;

int final_ret;
int borderCols;
int smallBlockCol;
int blockCols;
int *gpuWall, *gpuResult[2];
#define BLOCK_SIZE 256

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )

extern t_tqueue *tqueues;

 __device__ uint get_smid_PF(void) {
	uint ret;

	asm("mov.u32 %0, %smid;" : "=r"(ret) );

	return ret;
}


/**
 * Path Finder (CUDA Kernel)
 */
__global__ void
original_pathFinderCUDA(int pyramid_heightPF, int *gpuWall, int *gpuSrc, int *gpuResults, int cols,  int rows, int startStep, int border)
{
    for(startStep = 0; startStep < rows - 1; startStep += pyramid_heightPF){
	
		int iteration = MIN(pyramid_heightPF, rows-startStep-1);
		
		__shared__ int prev[BLOCK_SIZE];
		__shared__ int result[BLOCK_SIZE];

		int bx = blockIdx.x;
		int tx = threadIdx.x;

		// each block finally computes result for a small block
		// after N iterations. 
		// it is the non-overlapping small blocks that cover 
		// all the input data

		// calculate the small block size
		int small_block_cols = BLOCK_SIZE-iteration*HALO*2;

		// calculate the boundary for the block according to 
		// the boundary of its small block
		int blkX = small_block_cols*bx-border;
		int blkXmax = blkX+BLOCK_SIZE-1;

		// calculate the global thread coordination
		int xidx = blkX+tx;

		// effective range within this block that falls within 
		// the valid range of the input data
		// used to rule out computation outside the boundary.
		int validXmin = (blkX < 0) ? -blkX : 0;
		int validXmax = (blkXmax > cols-1) ? BLOCK_SIZE-1-(blkXmax-cols+1) : BLOCK_SIZE-1;

		int W = tx-1;
		int E = tx+1;

		W = (W < validXmin) ? validXmin : W;
		E = (E > validXmax) ? validXmax : E;

		bool isValid = IN_RANGE(tx, validXmin, validXmax);

		if(IN_RANGE(xidx, 0, cols-1)){
			prev[tx] = gpuSrc[xidx];
		}
		__syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
		bool computed;
		for (int i=0; i<iteration ; i++){ 
			computed = false;
			if( IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) &&  \
			isValid){
				computed = true;
				int left = prev[W];
				int up = prev[tx];
				int right = prev[E];
				int shortest = MIN(left, up);
				shortest = MIN(shortest, right);
				int index = cols*(startStep+i)+xidx;
				result[tx] = shortest + gpuWall[index];
			}
			__syncthreads();
			if(i==iteration-1)
				break;
			if(computed)   //Assign the computation range
				prev[tx]= result[tx];
			__syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
		}

		// update the global memory
		// after the last iteration, only threads coordinated within the 
		// small block perform the calculation and switch on ``computed''
		if (computed){
			gpuResults[xidx]=result[tx];    
		}
	}
}

__global__ void
SMT_pathFinderCUDA(int pyramid_heightPF, int *gpuWall, int *gpuSrc, int *gpuResults, int cols,  int rows, int startStep, int border,
					int SIMD_min, int SIMD_max,
					int num_subtask, int iter_per_subtask, int *cont_subtask, State *status)
{
	
	__shared__ int s_bid;
	
	unsigned int SM_id = get_smid_PF();
	
	if (SM_id <SIMD_min || SM_id > SIMD_max) /* Only blocks executing within SIMD_min and SIMD_max can progress */ 
			return;
	
	while (1){
		
		/********** Task Id calculation *************/
		
		if (threadIdx.x == 0) { 
			if (*status == TOEVICT)
				s_bid = -1;
			else {
				s_bid = atomicAdd(cont_subtask, 1);				//subtask_id
				//printf("Blq=%d cont=%d\n", blockIdx.x, s_bid);
			}
		}
		
		__syncthreads();
		
		//if (s_bid[warpid] >= num_subtask || s_bid[warpid] == -1)
		if (s_bid >=num_subtask || s_bid ==-1) /* If all subtasks have been executed */{
			//if (threadIdx.x == 0)  printf("El bloque %d se sale con %d\n", blockIdx.x, s_bid); 
			return;
		}

		for(startStep = 0; startStep < rows - 1; startStep += pyramid_heightPF){	
			int iteration = MIN(pyramid_heightPF, rows-startStep-1);
			
			__shared__ int prev[BLOCK_SIZE];
			__shared__ int result[BLOCK_SIZE];

			//int bx = blockIdx.x;
			int bx = s_bid;
			int tx = threadIdx.x;

			// each block finally computes result for a small block
			// after N iterations. 
			// it is the non-overlapping small blocks that cover 
			// all the input data

			// calculate the small block size
			int small_block_cols = BLOCK_SIZE-iteration*HALO*2;

			// calculate the boundary for the block according to 
			// the boundary of its small block
			int blkX = small_block_cols*bx-border;
			int blkXmax = blkX+BLOCK_SIZE-1;

			// calculate the global thread coordination
			int xidx = blkX+tx;

			// effective range within this block that falls within 
			// the valid range of the input data
			// used to rule out computation outside the boundary.
			int validXmin = (blkX < 0) ? -blkX : 0;
			int validXmax = (blkXmax > cols-1) ? BLOCK_SIZE-1-(blkXmax-cols+1) : BLOCK_SIZE-1;

			int W = tx-1;
			int E = tx+1;

			W = (W < validXmin) ? validXmin : W;
			E = (E > validXmax) ? validXmax : E;

			bool isValid = IN_RANGE(tx, validXmin, validXmax);

			if(IN_RANGE(xidx, 0, cols-1)){
				prev[tx] = gpuSrc[xidx];
			}
			__syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
			bool computed;
			for (int i=0; i<iteration ; i++){ 
				computed = false;
				if( IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) &&  \
				isValid){
					computed = true;
					int left = prev[W];
					int up = prev[tx];
					int right = prev[E];
					int shortest = MIN(left, up);
					shortest = MIN(shortest, right);
					int index = cols*(startStep+i)+xidx;
					result[tx] = shortest + gpuWall[index];
				}
				__syncthreads();
				if(i==iteration-1)
					break;
				if(computed)   //Assign the computation range
					prev[tx]= result[tx];
				__syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
			}

			// update the global memory
			// after the last iteration, only threads coordinated within the 
			// small block perform the calculation and switch on ``computed''
			if (computed){
				gpuResults[xidx]=result[tx];    
			}
		}
	}
}

__global__ void
SMK_pathFinderCUDA(int pyramid_heightPF, int *gpuWall, int *gpuSrc, int *gpuResults, int cols,  int rows, int startStep, int border,
					int max_blocks_per_SM,
					int num_subtask,
					int iter_per_subtask,
					int *cont_SM,
					int *cont_subtask,
					State *status
)
{
	__shared__ int s_bid, s_index;
	
	unsigned int SM_id = get_smid_PF();
	
	if (threadIdx.x == 0)  
		s_index = atomicAdd(&cont_SM[SM_id],1);
	
	__syncthreads();

	if (s_index > max_blocks_per_SM)
		return;
	
	while (1){
		
		/********** Task Id calculation *************/
		if (threadIdx.x == 0) {
			if (*status == TOEVICT)
				s_bid = -1;
			else
				s_bid = atomicAdd(cont_subtask, 1);
		}
		
		__syncthreads();
		
		if (s_bid >= num_subtask || s_bid == -1) /* If all subtasks have been executed */
			return;
		
		for(startStep = 0; startStep < rows - 1; startStep += pyramid_heightPF){	
			int iteration = MIN(pyramid_heightPF, rows-startStep-1);
			
			__shared__ int prev[BLOCK_SIZE];
			__shared__ int result[BLOCK_SIZE];

			int bx = s_bid;
			int tx = threadIdx.x;

			// each block finally computes result for a small block
			// after N iterations. 
			// it is the non-overlapping small blocks that cover 
			// all the input data

			// calculate the small block size
			int small_block_cols = BLOCK_SIZE-iteration*HALO*2;

			// calculate the boundary for the block according to 
			// the boundary of its small block
			int blkX = small_block_cols*bx-border;
			int blkXmax = blkX+BLOCK_SIZE-1;

			// calculate the global thread coordination
			int xidx = blkX+tx;

			// effective range within this block that falls within 
			// the valid range of the input data
			// used to rule out computation outside the boundary.
			int validXmin = (blkX < 0) ? -blkX : 0;
			int validXmax = (blkXmax > cols-1) ? BLOCK_SIZE-1-(blkXmax-cols+1) : BLOCK_SIZE-1;

			int W = tx-1;
			int E = tx+1;

			W = (W < validXmin) ? validXmin : W;
			E = (E > validXmax) ? validXmax : E;

			bool isValid = IN_RANGE(tx, validXmin, validXmax);

			if(IN_RANGE(xidx, 0, cols-1)){
				prev[tx] = gpuSrc[xidx];
			}
			__syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
			bool computed;
			for (int i=0; i<iteration ; i++){ 
				computed = false;
				if( IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) &&  \
				isValid){
					computed = true;
					int left = prev[W];
					int up = prev[tx];
					int right = prev[E];
					int shortest = MIN(left, up);
					shortest = MIN(shortest, right);
					int index = cols*(startStep+i)+xidx;
					result[tx] = shortest + gpuWall[index];
				}
				__syncthreads();
				if(i==iteration-1)
					break;
				if(computed)   //Assign the computation range
					prev[tx]= result[tx];
				__syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
			}

			// update the global memory
			// after the last iteration, only threads coordinated within the 
			// small block perform the calculation and switch on ``computed''
			if (computed){
				gpuResults[xidx]=result[tx];    
			}
		}
	}
}

void
init()
{
	data = new int[rows*cols];
	wall = new int*[rows];
	for(int n=0; n<rows; n++)
		wall[n]=data+cols*n;
	result = new int[cols];
	
	int seed = M_SEED;
	srand(seed);

	for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            wall[i][j] = rand() % 10;
        }
    }

	// printf("WALL\n");
    // for (int i = 0; i < rows; i++)
    // {
        // for (int j = 0; j < cols; j++)
        // {
            // printf("%d ",wall[i][j]) ;
        // }
        // printf("\n") ;
    // }

}

int PF_start_kernel(void *arg) 
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	int blocksize = kstub->kconf.blocksize.x;
	
	t_PF_params * params = (t_PF_params *)kstub->params;
	
	cols = params->nCols;
	rows = params->nRows;
	pyramid_height = params->param_pyramid_height;
	
	data = new int[rows*cols];
	wall = new int*[rows];
	result = new int[cols];
	
	init();

	/* --------------- pyramid parameters --------------- */
    borderCols = (pyramid_height)*HALO;
    smallBlockCol = blocksize-(pyramid_height)*HALO*2;
    blockCols = cols/smallBlockCol+((cols%smallBlockCol==0)?0:1);

    // printf("pyramidHeight: %d\ngridSize: [%d]\nborder:[%d]\nblockSize: %d\nblockGrid:[%d]\ntargetBlock:[%d]\n",\
	// pyramid_height, cols, borderCols, blocksize, blockCols, smallBlockCol);
	
    int size = rows*cols;

    hipMalloc((void**)&gpuResult[0], sizeof(int)*cols);
    hipMalloc((void**)&gpuResult[1], sizeof(int)*cols);
    hipMemcpy(gpuResult[0], data, sizeof(int)*cols, hipMemcpyHostToDevice);
    hipMalloc((void**)&gpuWall, sizeof(int)*(size-cols));
    hipMemcpy(gpuWall, data+cols, sizeof(int)*(size-cols), hipMemcpyHostToDevice);
	
	return 0;
}

int PF_start_mallocs(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	int blocksize = kstub->kconf.blocksize.x;
	
	t_PF_params * params = (t_PF_params *)kstub->params;
	
	cols = params->nCols;
	rows = params->nRows;
	pyramid_height = params->param_pyramid_height;

	/* --------------- pyramid parameters --------------- */
    borderCols = (pyramid_height)*HALO;
    smallBlockCol = blocksize-(pyramid_height)*HALO*2;
    blockCols = cols/smallBlockCol+((cols%smallBlockCol==0)?0:1);

    // printf("pyramidHeight: %d\ngridSize: [%d]\nborder:[%d]\nblockSize: %d\nblockGrid:[%d]\ntargetBlock:[%d]\n",\
	// pyramid_height, cols, borderCols, blocksize, blockCols, smallBlockCol);
	
    int size = rows*cols;
	
#if defined(MEMCPY_SYNC) || defined(MEMCPY_ASYNC)
	hipHostMalloc(&data, sizeof(int)*(rows*cols));
	hipHostMalloc(&wall, sizeof(int)*rows);
	hipHostMalloc(&result, sizeof(int)*cols);

	hipMalloc((void**)&gpuResult[0], sizeof(int)*cols);
    hipMalloc((void**)&gpuResult[1], sizeof(int)*cols);
	hipMalloc((void**)&gpuWall, sizeof(int)*(size-cols));
#else
	#ifdef MANAGED_MEM

	hipMallocManaged(&data, sizeof(int)*(rows, cols));
	hipMallocManaged(&wall, sizeof(int)*rows);
	hipMallocManaged(&result, sizeof(int)*cols);
	
	gpuResult[0] = data;
	gpuWall = data+cols;
	#else
		printf("No transfer model: Exiting ...\n");
		exit(-1);
	#endif
#endif

	// Verify that allocations succeeded
    if (data == NULL || wall == NULL || result == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    init();

	return 0;
}

int PF_start_transfers(void *arg)
{
	hipError_t err = hipSuccess;
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	
	int size = rows*cols;
	
#ifdef MEMCPY_SYNC
	enqueue_tcomamnd(tqueues, gpuResult[0], data, sizeof(int)*cols, hipMemcpyHostToDevice, 0, BLOCKING, DATA, LOW, kstub);

	enqueue_tcomamnd(tqueues, gpuWall, data+cols, sizeof(int)*(size-cols), hipMemcpyHostToDevice, 0, BLOCKING, DATA, LOW, kstub);
	
	kstub->HtD_tranfers_finished = 1;

	
#else
	
	#ifdef MEMCPY_ASYNC
	
	//enqueue_tcomamnd(tqueues, gpuResult[0], data, sizeof(int)*cols, hipMemcpyHostToDevice, 0, NONBLOCKING, DATA, MEDIUM, kstub);
	//enqueue_tcomamnd(tqueues, gpuWall, data+cols, sizeof(int)*(size-cols), hipMemcpyHostToDevice, 0, NONBLOCKING, LAST_TRANSFER, MEDIUM, kstub);

	 err = hipMemcpyAsync(gpuResult[0], data, sizeof(int)*cols, hipMemcpyHostToDevice, kstub->transfer_s[0]);
	 err = hipMemcpyAsync(gpuWall, data+cols, sizeof(int)*(size-cols), hipMemcpyHostToDevice, kstub->transfer_s[0]);
	
	#else
	#ifdef MANAGED_MEM

	hipDeviceProp_t p;
    hipGetDeviceProperties(&p, kstub->deviceId);
	
	if (p.concurrentManagedAccess)
	{
		err = hipMemPrefetchAsync(data, sizeof(int)*(rows, cols), kstub->deviceId);
		if ( err != hipSuccess) {
			printf("Error in vAdd:hipMemPrefetchAsync\n");
			exit(EXIT_FAILURE);
		}
		err = hipMemPrefetchAsync(wall, sizeof(int)*rows, kstub->deviceId);
		if ( err != hipSuccess) {
			printf("Error in vAdd:hipMemPrefetchAsync\n");
			exit(EXIT_FAILURE);
		}
		err = hipMemPrefetchAsync(result, sizeof(int)*cols, kstub->deviceId);
		if ( err != hipSuccess) {
			printf("Error in vAdd:hipMemPrefetchAsync\n");
			exit(EXIT_FAILURE);
		}
	}
	//hipEventRecord(kstub->end_HtD, kstub->transfer_s[0]);
	
	//hipStreamSynchronize(kstub->transfer_s[0]);
	kstub->HtD_tranfers_finished = 1;

	#endif
	#endif
#endif

	return 0;
}

// int PF_end_kernel_dummy(void *arg)
// {
	// hipMemcpy(result, gpuResult[final_ret], sizeof(int)*cols, hipMemcpyDeviceToHost);
	
	
	// hipFree(gpuWall);
    // hipFree(gpuResult[0]);
    // hipFree(gpuResult[1]);

    // hipHostFree(data);
	// hipHostFree(wall);
	// hipHostFree(result);

    // return 0;
// }

int PF_end_kernel(void *arg)
{
	
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	
#ifdef MEMCPY_SYNC

	hipEventSynchronize(kstub->end_Exec);

	enqueue_tcomamnd(tqueues, result, gpuResult[final_ret], sizeof(int)*cols, hipMemcpyDeviceToHost, 0, BLOCKING, DATA, LOW, kstub);
	 
#else
	#ifdef MEMCPY_ASYNC
	printf("-->Comienzo de DtH para tarea %d\n", kstub->id);

	//enqueue_tcomamnd(tqueues, result, gpuResult[final_ret], sizeof(int)*cols, hipMemcpyDeviceToHost, kstub->transfer_s[1] , NONBLOCKING, LAST_TRANSFER, MEDIUM, kstub);
	
	hipMemcpyAsync(result, gpuResult[final_ret], sizeof(int)*cols, hipMemcpyDeviceToHost, kstub->transfer_s[1]);
	#else
		#ifdef MANAGED_MEM
			hipStreamSynchronize(*(kstub->execution_s)); // To be sure kernel execution has finished before processing output data
		#endif
	#endif
#endif

	return 0;
}	
 
int launch_orig_PF(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	
	// Setup execution parameters
    //dim3 threads(kstub->kconf.blocksize.x, kstub->kconf.blocksize.y);
	//dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid(blockCols);
	
	int src = 1, dst = 0, t = 0;
	
	int temp = src;
	src = dst;
	dst = temp;
	
	original_pathFinderCUDA<<<kstub->kconf.gridsize.x, kstub->kconf.blocksize.x>>>(
		pyramid_height, 
		gpuWall, gpuResult[src], gpuResult[dst],
		cols,rows, t, borderCols);
		
	// for the measurement fairness
	//hipDeviceSynchronize();
	
	final_ret = dst;

	return 0;
}

int launch_preemp_PF(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	
	// Setup execution parameters
    //dim3 threads(kstub->kconf.blocksize.x, kstub->kconf.blocksize.y);
	//dim3 dimBlock(BLOCK_SIZE);
	//dim3 dimGrid(blockCols);
	
    int src = 1, dst = 0, t = 0;
	
	int temp = src;
	src = dst;
	dst = temp;
	
	#ifdef SMT
		SMT_pathFinderCUDA<<< kstub->kconf.numSMs * kstub->kconf.max_persistent_blocks, kstub->kconf.blocksize.x, 0, *(kstub->execution_s) >>>(
			pyramid_height, 
			gpuWall, gpuResult[src], gpuResult[dst],
			cols,rows, t, borderCols,
			
			kstub->idSMs[0],
			kstub->idSMs[1],
			kstub->total_tasks,
			kstub->kconf.coarsening,
			kstub->d_executed_tasks,
			&(kstub->gm_state[kstub->stream_index]));
	#else
		SMK_pathFinderCUDA<<< kstub->kconf.numSMs * kstub->kconf.max_persistent_blocks, kstub->kconf.blocksize.x, 0, *(kstub->execution_s) >>>(
			pyramid_height, 
			gpuWall, gpuResult[src], gpuResult[dst],
			cols,rows, t, borderCols,
			
			kstub->num_blocks_per_SM,
			kstub->total_tasks,
			kstub->kconf.coarsening,
			kstub->d_SMs_cont,
			kstub->d_executed_tasks,
			&(kstub->gm_state[kstub->stream_index]));
	#endif

	// for the measurement fairness
	//hipDeviceSynchronize();
	
	final_ret = dst;
	
	return 0;
}
 



 