#include <unistd.h>
#include <time.h>
#include <string.h>
#include <unistd.h>
#include <helper_functions.h>   // helper functions for string parsing
#include <hip/hip_runtime_api.h>   
#include "elastic_kernel.h"

t_coBlocks info_coBlocks[Number_of_Kernels-1][Number_of_Kernels-1];

t_solo info_solo[Number_of_Kernels-1];

t_coBlocks *fill_head(t_Kernel k1, t_Kernel k2, int num_configs)
{
	t_coBlocks *myinfo;
	
	myinfo = &info_coBlocks[k1][k2];
	myinfo->kid[0] = k1; myinfo->kid[1] = k2; 
	myinfo->num_configs = num_configs;
	
	myinfo->pairs = (int **) calloc(myinfo->num_configs, sizeof(int *));
	myinfo->tpms = (double **) calloc(myinfo->num_configs, sizeof(double *));
	for (int i=0; i < myinfo->num_configs; i++) {
		myinfo->pairs[i] = (int *)calloc(2, sizeof(int));
		myinfo->tpms[i] = (double *)calloc(2, sizeof(double));
	}
	
	return myinfo;
}

int reverse_values(t_coBlocks *info)
{
	t_coBlocks *new_info = fill_head(info->kid[1], info->kid[0], info->num_configs);
	
	for (int i=0; i < info->num_configs; i++) {
		new_info->pairs[i][0] = info->pairs[i][1];
		new_info->pairs[i][1] = info->pairs[i][0];
	}
		
	return 0;
}

int fill_coBlocks()
{
	
	memset (info_coBlocks, 0, (Number_of_Kernels-1) * (Number_of_Kernels-1) * sizeof(t_coBlocks));
	t_coBlocks *myinfo, *save_info;
	
	//MM-BS
	/*myinfo = &info_coBlocks[MM][BS];
	myinfo->kid[0] = MM; myinfo->kid[1] = BS; 
	myinfo->num_configs = 7;
	
	myinfo->pairs = (int **) calloc(myinfo->num_configs, sizeof(int *));
	for (int i=0; i < myinfo->num_configs; i++)
		myinfo->pairs[i] = (int *)calloc(2, sizeof(int));
	*/
	
	myinfo = fill_head(MM, BS, 7);

	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 7;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 6;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 5;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 4;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 3;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	
	reverse_values(myinfo);

	save_info = myinfo;
	
	//MM-VA
	
	myinfo = fill_head(MM, VA, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	
	reverse_values(myinfo);
	
	//MM-Reduction
	
	myinfo = fill_head(MM, Reduction, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	
	reverse_values(myinfo);

	//MM-PF
	
	myinfo = fill_head(MM, PF, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//MM-GCEDD
	
	myinfo = fill_head(MM, GCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//MM-SCEDD
	
	myinfo = fill_head(MM, SCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//MM-NCEDD
	
	myinfo = fill_head(MM, NCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//MM-HCEDD
	
	myinfo = fill_head(MM, HCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//MM-SPMV_CSRscalar
	
	myinfo = fill_head(MM, SPMV_CSRscalar, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 14;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 12;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 10;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 8;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 6;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 4;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 2;
	reverse_values(myinfo);
	
	//MM-HST256
	
	myinfo = fill_head(MM, HST256, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 9;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 8;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 6;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 5;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 4;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	
	//MM-RCONV
	
	myinfo = fill_head(MM, RCONV, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 25;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 24;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 20;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 16;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 12;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 8;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 4;
	reverse_values(myinfo);
	
	/////////////////////////////////////////////////////////////////
	
	//BS-VA
	
	myinfo = fill_head(BS, VA, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	
	reverse_values(myinfo);
	
	//BS-Reduction
	
	myinfo = fill_head(BS, Reduction, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//BS-PF
	
	myinfo = fill_head(BS, PF, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//BS-GCEDD
	
	myinfo = fill_head(BS, GCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//BS-SCEDD
	
	myinfo = fill_head(BS, SCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//BS-NCEDD
	
	myinfo = fill_head(BS, NCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//BS-HCEDD
	
	myinfo = fill_head(BS, HCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	
	//BS-SPMV_CSRscalar
	
	myinfo = fill_head(BS, SPMV_CSRscalar, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 14;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 12;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 10;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 8;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 6;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 4;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 2;
	reverse_values(myinfo);
	
	//BS-HST256
	
	myinfo = fill_head(BS, HST256, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 9;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 8;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 6;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 5;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 4;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	
	//BS-RCONV
	
	myinfo = fill_head(BS, RCONV, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 26;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 24;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 20;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 16;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 12;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 8;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 4;
	reverse_values(myinfo);
	
	////////////////////////////////////////////////////////////////////////////////////////
	
	//VA-Reduction
	
	myinfo = fill_head(VA, Reduction, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//VA-PF
	
	myinfo = fill_head(VA, PF, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//VA-GCEDD
	
	myinfo = fill_head(VA, GCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//MM-SCEDD
	
	myinfo = fill_head(VA, SCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//MM-NCEDD
	
	myinfo = fill_head(VA, NCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//MM-HCEDD
	
	myinfo = fill_head(VA, HCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//VA-SPMV_CSRscalar
	
	myinfo = fill_head(VA, SPMV_CSRscalar, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 14;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 12;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 10;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 8;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 6;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 4;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 2;
	reverse_values(myinfo);
	
	//VA-HST256
	
	myinfo = fill_head(VA, HST256, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 9;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 8;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 6;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 5;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 4;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	
	//VA-RCONV
	
	myinfo = fill_head(VA, RCONV, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 26;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 24;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 20;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 16;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 12;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 8;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 4;
	reverse_values(myinfo);
	
	//////////////////////////////////////////////////////////////
	
	//SPMV_CSRscalar -Reduction
	
	myinfo = fill_head(SPMV_CSRscalar, Reduction, 7);
	
	myinfo->pairs[0][0] = 2; myinfo->pairs[0][1] = 7;
	myinfo->pairs[1][0] = 4; myinfo->pairs[1][1] = 6;
	myinfo->pairs[2][0] = 6; myinfo->pairs[2][1] = 5;
	myinfo->pairs[3][0] = 8; myinfo->pairs[3][1] = 4;
	myinfo->pairs[4][0] = 10; myinfo->pairs[4][1] = 3;
	myinfo->pairs[5][0] = 12; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 14; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	
	save_info = myinfo; 
	
	//SPMV_CSRscalar - PF
	
	myinfo = fill_head(SPMV_CSRscalar, PF, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//SPMV_CSRscalar - GCEDD
	
	myinfo = fill_head(SPMV_CSRscalar, GCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//SPMV_CSRscalar-SCEDD
	
	myinfo = fill_head(SPMV_CSRscalar, SCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//SPMV_CSRscalar-NCEDD
	
	myinfo = fill_head(SPMV_CSRscalar, NCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//SPMV_CSRscalar-HCEDD
	
	myinfo = fill_head(SPMV_CSRscalar, HCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//SPMV_CSRscalar - RCONV
	
	myinfo = fill_head(SPMV_CSRscalar, RCONV, 13);
	
	
	myinfo->pairs[0][0] = 3; myinfo->pairs[0][1] = 26;
	myinfo->pairs[1][0] = 4; myinfo->pairs[1][1] = 24;
	myinfo->pairs[2][0] = 5; myinfo->pairs[2][1] = 22;
	myinfo->pairs[3][0] = 6; myinfo->pairs[3][1] = 20;
	myinfo->pairs[4][0] = 7; myinfo->pairs[4][1] = 18;
	myinfo->pairs[5][0] = 8; myinfo->pairs[5][1] = 16;
	myinfo->pairs[6][0] = 9; myinfo->pairs[6][1] = 14;
	myinfo->pairs[7][0] = 10; myinfo->pairs[7][1] = 12;
	myinfo->pairs[8][0] = 11; myinfo->pairs[8][1] = 10;
	myinfo->pairs[9][0] = 12; myinfo->pairs[9][1] = 8;
	myinfo->pairs[10][0] = 13; myinfo->pairs[10][1] = 6;
	myinfo->pairs[11][0] = 14; myinfo->pairs[11][1] = 4;
	myinfo->pairs[12][0] = 15; myinfo->pairs[12][1] = 2;
	reverse_values(myinfo);
	
	//SPMV_CSRscalar - HST256
	
	myinfo = fill_head(SPMV_CSRscalar, HST256, 10);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 10;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 9;
	myinfo->pairs[2][0] = 4; myinfo->pairs[2][1] = 8;
	myinfo->pairs[3][0] = 5; myinfo->pairs[3][1] = 7;
	myinfo->pairs[4][0] = 7; myinfo->pairs[4][1] = 6;
	myinfo->pairs[5][0] = 8; myinfo->pairs[5][1] = 5;
	myinfo->pairs[6][0] = 10; myinfo->pairs[6][1] = 4;
	myinfo->pairs[7][0] = 11; myinfo->pairs[7][1] = 3;
	myinfo->pairs[8][0] = 12; myinfo->pairs[8][1] = 2;
	myinfo->pairs[9][0] = 14; myinfo->pairs[9][1] = 1;
	reverse_values(myinfo);
	
	////////////////////////////////////////////////////////////////
	
	//Reduction - PF
	
	myinfo = fill_head(Reduction, PF, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 7;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 6;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 5;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 4;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 3;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	save_info = myinfo;
	reverse_values(myinfo);

	//Reduction - GCEDD
	
	myinfo = fill_head(Reduction, GCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//Reduction-SCEDD
	
	myinfo = fill_head(Reduction, SCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//Reduction-NCEDD
	
	myinfo = fill_head(Reduction, NCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//Reduction-HCEDD
	
	myinfo = fill_head(Reduction, HCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);

	// Reduction - RCONV 
	
	myinfo = fill_head(Reduction, RCONV, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 26;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 24;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 20;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 16;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 12;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 8;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 4;
	reverse_values(myinfo);
	
	// Reduction -  HST256
	
	myinfo = fill_head(Reduction, HST256, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 9;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 8;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 6;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 5;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 4;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	
	////////////////////////////////////////////////////////////////////////////////
	
	// PF - GCEDD
	
	myinfo = fill_head(PF, GCEDD, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 7;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 6;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 5;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 4;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 3;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	save_info = myinfo;
	
	//PF-SCEDD
	
	myinfo = fill_head(PF, SCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//PF-NCEDD
	
	myinfo = fill_head(PF, NCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//PF-HCEDD
	
	myinfo = fill_head(PF, HCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	// PF - RCONV
	
	myinfo = fill_head(PF, RCONV, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 26;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 24;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 20;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 16;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 12;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 8;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 4;
	reverse_values(myinfo);
	
	// PF - HST256
	
	myinfo = fill_head(PF, HST256, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 9;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 8;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 6;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 5;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 4;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	
	//////////////////////////////////////////////////////////
	
	// RCONV-GCEDD
	
	myinfo = fill_head(RCONV, GCEDD, 7);
	
	myinfo->pairs[0][0] = 4; myinfo->pairs[0][1] = 7;
	myinfo->pairs[1][0] = 8; myinfo->pairs[1][1] = 6;
	myinfo->pairs[2][0] = 12; myinfo->pairs[2][1] = 5;
	myinfo->pairs[3][0] = 16; myinfo->pairs[3][1] = 4;
	myinfo->pairs[4][0] = 20; myinfo->pairs[4][1] = 3;
	myinfo->pairs[5][0] = 24; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 25; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	save_info = myinfo;
	
	//RCONV-SCEDD
	
	myinfo = fill_head(RCONV, SCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//RCONV-NCEDD
	
	myinfo = fill_head(RCONV, NCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//RCONV-HCEDD
	
	myinfo = fill_head(RCONV, HCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	// RCONV - HST256
	
	myinfo = fill_head(RCONV, HST256, 10);
	
	myinfo->pairs[0][0] = 2; myinfo->pairs[0][1] = 10;
	myinfo->pairs[1][0] = 4; myinfo->pairs[1][1] = 9;
	myinfo->pairs[2][0] = 6; myinfo->pairs[2][1] = 8;
	myinfo->pairs[3][0] = 9; myinfo->pairs[3][1] = 7;
	myinfo->pairs[4][0] = 11; myinfo->pairs[4][1] = 6;
	myinfo->pairs[5][0] = 14; myinfo->pairs[5][1] = 5;
	myinfo->pairs[6][0] = 16; myinfo->pairs[6][1] = 4;
	myinfo->pairs[7][0] = 18; myinfo->pairs[7][1] = 3;
	myinfo->pairs[8][0] = 21; myinfo->pairs[8][1] = 2;
	myinfo->pairs[9][0] = 23; myinfo->pairs[9][1] = 1;
	
	reverse_values(myinfo);

	////////////////////////////////////////////////////////////
		
	// GCEDD - HST256
	
	myinfo = fill_head(GCEDD, HST256, 7);

	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 9;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 8;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 6;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 5;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 4;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	save_info = myinfo;
	
	//GCEDD-SCEDD
	
	myinfo = fill_head(GCEDD, SCEDD, 7);

	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 7;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 6;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 5;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 4;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 3;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	save_info = myinfo;
	
	//GCEDD-NCEDD
	
	myinfo = fill_head(GCEDD, NCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//GCEDD-HCEDD
	
	myinfo = fill_head(GCEDD, HCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	/////////////////////////////////////////////7777
	
	//HST256 - SCEDD
	
	myinfo = fill_head(HST256, SCEDD, 7);

	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 7;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 6;
	myinfo->pairs[2][0] = 4; myinfo->pairs[2][1] = 5;
	myinfo->pairs[3][0] = 5; myinfo->pairs[3][1] = 4;
	myinfo->pairs[4][0] = 6; myinfo->pairs[4][1] = 3;
	myinfo->pairs[5][0] = 8; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 9; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	save_info = myinfo;
	
	//HST256-NCEDD
	
	myinfo = fill_head(HST256, NCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//HST256-HCEDD
	
	myinfo = fill_head(HST256, HCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	///////////////////////////////////////////////////////
	
	//SCEDD-NCEDD
	
	myinfo = fill_head(SCEDD, NCEDD, 7);

	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 7;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 6;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 5;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 4;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 3;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	save_info = myinfo;
	
	//SCEDD-HCEDD
	
	myinfo = fill_head(SCEDD, HCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	/////////////////////////////////////////
	
	// NCEDD - HCEDD
	
	myinfo = fill_head(NCEDD, HCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	/////////////////////////////////////////////////////////////
	
	// CCONV - MM
	myinfo = fill_head(CCONV, MM, 6);

	myinfo->pairs[0][0] = 2; myinfo->pairs[0][1] = 7;
	myinfo->pairs[1][0] = 4; myinfo->pairs[1][1] = 6;
	myinfo->pairs[2][0] = 6; myinfo->pairs[2][1] = 5;
	myinfo->pairs[3][0] = 8; myinfo->pairs[3][1] = 4;
	myinfo->pairs[4][0] = 10; myinfo->pairs[4][1] = 3;
	myinfo->pairs[5][0] = 11; myinfo->pairs[5][1] = 2;
	reverse_values(myinfo);
	save_info = myinfo;
	
	// CCONV - BS
	
	myinfo = fill_head(CCONV, BS, 6);
	
	for (int i=0; i<6;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	// CCONV - VA
	
	myinfo = fill_head(CCONV, VA, 6);
	
	for (int i=0; i<6;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	// CCONV - PF

	myinfo = fill_head(CCONV, PF, 6);
	
	for (int i=0; i<6;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);	
	
	// CCONV - Reduction
	 
	
	for (int i=0; i<6;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);	
	
	// CCONV - GCEDD
	
	myinfo = fill_head(CCONV, GCEDD, 6);
	
	for (int i=0; i<6;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);	
	
	//CCONV - SCEDD
	
	myinfo = fill_head(CCONV, SCEDD, 6);
	
	for (int i=0; i<6;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);	
	
	//CCONV - NCEDD
	
	myinfo = fill_head(CCONV, NCEDD, 6);
	
	for (int i=0; i<6;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);	
	
	//CCONV - HCEDD
	
	myinfo = fill_head(CCONV, HCEDD, 6);
	
	for (int i=0; i<6;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);	 
	
	//CCONV - SPMV_CSRscalar
	
	myinfo = fill_head(CCONV, SPMV_CSRscalar, 11);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 15;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 14;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 13;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 12;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 11;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 10;
	myinfo->pairs[6][0] = 7; myinfo->pairs[0][1] = 9;
	myinfo->pairs[7][0] = 8; myinfo->pairs[1][1] = 8;
	myinfo->pairs[8][0] = 9; myinfo->pairs[2][1] = 7;
	myinfo->pairs[9][0] = 10; myinfo->pairs[3][1] = 6;
	myinfo->pairs[10][0] = 11; myinfo->pairs[4][1] = 5;
	reverse_values(myinfo);	
	
	//CCONV - RCONV
	
	myinfo = fill_head(CCONV, RCONV, 11);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 24;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 22;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 20;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 18;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 16;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 14;
	myinfo->pairs[6][0] = 7; myinfo->pairs[0][1] = 12;
	myinfo->pairs[7][0] = 8; myinfo->pairs[1][1] = 9;
	myinfo->pairs[8][0] = 9; myinfo->pairs[2][1] = 7;
	myinfo->pairs[9][0] = 10; myinfo->pairs[3][1] = 5;
	myinfo->pairs[10][0] = 11; myinfo->pairs[4][1] = 3;
	reverse_values(myinfo);	
	
	
	//CCONV - HST256
	
	myinfo = fill_head(CCONV, HST256, 11);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 10;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 9;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 8;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 7;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 6;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 5;
	myinfo->pairs[6][0] = 7; myinfo->pairs[0][1] = 5;
	myinfo->pairs[7][0] = 8; myinfo->pairs[1][1] = 4;
	myinfo->pairs[8][0] = 9; myinfo->pairs[2][1] = 3;
	myinfo->pairs[9][0] = 10; myinfo->pairs[3][1] = 2;
	myinfo->pairs[10][0] = 11; myinfo->pairs[4][1] = 1;
	reverse_values(myinfo);	
	
	
	return 0;
}

int fill_solo()
{
	
	info_solo[MM].num_configs=8; //max_nm_blocks_per_SM
	info_solo[MM].tpms = (double *)calloc(info_solo[MM].num_configs, sizeof(double));
	
	info_solo[BS].num_configs=8; //max_nm_blocks_per_SM
	info_solo[BS].tpms = (double *)calloc(info_solo[BS].num_configs, sizeof(double));
	
	info_solo[VA].num_configs=8; //max_nm_blocks_per_SM
	info_solo[VA].tpms = (double *)calloc(info_solo[VA].num_configs, sizeof(double));
	
	info_solo[SPMV_CSRscalar].num_configs=16; //max_nm_blocks_per_SM
	info_solo[SPMV_CSRscalar].tpms = (double *)calloc(info_solo[SPMV_CSRscalar].num_configs, sizeof(double));
	
	info_solo[Reduction].num_configs=8; //max_nm_blocks_per_SM
	info_solo[Reduction].tpms = (double *)calloc(info_solo[Reduction].num_configs, sizeof(double));
	
	info_solo[PF].num_configs=8; //max_nm_blocks_per_SM
	info_solo[PF].tpms = (double *)calloc(info_solo[PF].num_configs, sizeof(double));
	
	info_solo[RCONV].num_configs=25; //max_nm_blocks_per_SM
	info_solo[RCONV].tpms = (double *)calloc(info_solo[RCONV].num_configs, sizeof(double));
	
	info_solo[GCEDD].num_configs=8; //max_nm_blocks_per_SM
	info_solo[GCEDD].tpms = (double *)calloc(info_solo[GCEDD].num_configs, sizeof(double));
	
	info_solo[HST256].num_configs=10; //max_nm_blocks_per_SM
	info_solo[HST256].tpms = (double *)calloc(info_solo[HST256].num_configs, sizeof(double));
	
	info_solo[SCEDD].num_configs=8; //max_nm_blocks_per_SM
	info_solo[SCEDD].tpms = (double *)calloc(info_solo[SCEDD].num_configs, sizeof(double));
	
	info_solo[NCEDD].num_configs=8; //max_nm_blocks_per_SM
	info_solo[NCEDD].tpms = (double *)calloc(info_solo[NCEDD].num_configs, sizeof(double));
	
	info_solo[HCEDD].num_configs=8; //max_nm_blocks_per_SM
	info_solo[HCEDD].tpms = (double *)calloc(info_solo[HCEDD].num_configs, sizeof(double));
	
	info_solo[CCONV].num_configs=12; //max_nm_blocks_per_SM
	info_solo[CCONV].tpms = (double *)calloc(info_solo[CCONV].num_configs, sizeof(double));
	
	
	
	return 0;
}
