#include <unistd.h>
#include <time.h>
#include <string.h>
#include <unistd.h>
#include <helper_functions.h>   // helper functions for string parsing
#include <hip/hip_runtime_api.h>   
#include "elastic_kernel.h"

t_coBlocks info_coBlocks[Number_of_Kernels-1][Number_of_Kernels-1];

t_solo info_solo[Number_of_Kernels-1];

t_coBlocks *fill_head(t_Kernel k1, t_Kernel k2, int num_configs)
{
	t_coBlocks *myinfo;
	
	myinfo = &info_coBlocks[k1][k2];
	myinfo->kid[0] = k1; myinfo->kid[1] = k2; 
	myinfo->num_configs = num_configs;
	
	myinfo->pairs = (int **) calloc(myinfo->num_configs, sizeof(int *));
	myinfo->tpms = (double **) calloc(myinfo->num_configs, sizeof(double *));
	for (int i=0; i < myinfo->num_configs; i++) {
		myinfo->pairs[i] = (int *)calloc(2, sizeof(int));
		myinfo->tpms[i] = (double *)calloc(2, sizeof(double));
	}
	
	return myinfo;
}

int reverse_values(t_coBlocks *info)
{
	t_coBlocks *new_info = fill_head(info->kid[1], info->kid[0], info->num_configs);
	
	for (int i=0; i < info->num_configs; i++) {
		new_info->pairs[i][0] = info->pairs[i][1];
		new_info->pairs[i][1] = info->pairs[i][0];
	}
		
	return 0;
}

int fill_coBlocks()
{
	
	memset (info_coBlocks, 0, (Number_of_Kernels-1) * (Number_of_Kernels-1) * sizeof(t_coBlocks));
	t_coBlocks *myinfo, *save_info;
	
	//MM-BS
	/*myinfo = &info_coBlocks[MM][BS];
	myinfo->kid[0] = MM; myinfo->kid[1] = BS; 
	myinfo->num_configs = 7;
	
	myinfo->pairs = (int **) calloc(myinfo->num_configs, sizeof(int *));
	for (int i=0; i < myinfo->num_configs; i++)
		myinfo->pairs[i] = (int *)calloc(2, sizeof(int));
	*/
	
	myinfo = fill_head(MM, BS, 7);

	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 7;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 6;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 5;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 4;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 3;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	
	reverse_values(myinfo);

	save_info = myinfo;
	
	//MM-VA
	
	myinfo = fill_head(MM, VA, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	
	reverse_values(myinfo);
	
	//MM-Reduction
	
	myinfo = fill_head(MM, Reduction, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	
	reverse_values(myinfo);

	//MM-PF
	
	myinfo = fill_head(MM, PF, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//MM-GCEDD
	
	myinfo = fill_head(MM, GCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//MM-SPMV_CSRscalar
	
	myinfo = fill_head(MM, SPMV_CSRscalar, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 14;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 12;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 10;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 8;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 6;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 4;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 2;
	reverse_values(myinfo);
	
	//MM-HST256
	
	myinfo = fill_head(MM, HST256, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 9;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 8;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 6;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 5;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 4;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	
	//MM-RCONV
	
	myinfo = fill_head(MM, RCONV, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 25;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 24;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 20;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 16;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 12;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 8;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 4;
	reverse_values(myinfo);
	
	/////////////////////////////////////////////////////////////////
	
	//BS-VA
	
	myinfo = fill_head(BS, VA, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	
	reverse_values(myinfo);
	
	//BS-Reduction
	
	myinfo = fill_head(BS, Reduction, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//BS-PF
	
	myinfo = fill_head(BS, PF, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//BS-GCEDD
	
	myinfo = fill_head(BS, GCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//BS-SPMV_CSRscalar
	
	myinfo = fill_head(BS, SPMV_CSRscalar, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 14;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 12;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 10;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 8;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 6;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 4;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 2;
	reverse_values(myinfo);
	
	//BS-HST256
	
	myinfo = fill_head(BS, HST256, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 9;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 8;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 6;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 5;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 4;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	
	//BS-RCONV
	
	myinfo = fill_head(BS, RCONV, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 26;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 24;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 20;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 16;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 12;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 8;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 4;
	reverse_values(myinfo);
	
	////////////////////////////////////////////////////////////////////////////////////////
	
	//VA-Reduction
	
	myinfo = fill_head(VA, Reduction, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//VA-PF
	
	myinfo = fill_head(VA, PF, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//VA-GCEDD
	
	myinfo = fill_head(VA, GCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//VA-SPMV_CSRscalar
	
	myinfo = fill_head(VA, SPMV_CSRscalar, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 14;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 12;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 10;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 8;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 6;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 4;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 2;
	reverse_values(myinfo);
	
	//VA-HST256
	
	myinfo = fill_head(VA, HST256, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 9;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 8;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 6;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 5;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 4;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	
	//VA-RCONV
	
	myinfo = fill_head(VA, RCONV, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 26;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 24;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 20;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 16;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 12;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 8;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 4;
	reverse_values(myinfo);
	
	//////////////////////////////////////////////////////////////
	
	//SPMV_CSRscalar -Reduction
	
	myinfo = fill_head(SPMV_CSRscalar, Reduction, 7);
	
	myinfo->pairs[0][0] = 2; myinfo->pairs[0][1] = 7;
	myinfo->pairs[1][0] = 4; myinfo->pairs[1][1] = 6;
	myinfo->pairs[2][0] = 6; myinfo->pairs[2][1] = 5;
	myinfo->pairs[3][0] = 8; myinfo->pairs[3][1] = 4;
	myinfo->pairs[4][0] = 10; myinfo->pairs[4][1] = 3;
	myinfo->pairs[5][0] = 12; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 14; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	
	save_info = myinfo; 
	
	//SPMV_CSRscalar - PF
	
	myinfo = fill_head(SPMV_CSRscalar, PF, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//SPMV_CSRscalar - GCEDD
	
	myinfo = fill_head(SPMV_CSRscalar, GCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);
	
	//SPMV_CSRscalar - RCONV
	
	myinfo = fill_head(SPMV_CSRscalar, RCONV, 13);
	
	
	myinfo->pairs[0][0] = 3; myinfo->pairs[0][1] = 26;
	myinfo->pairs[1][0] = 4; myinfo->pairs[1][1] = 24;
	myinfo->pairs[2][0] = 5; myinfo->pairs[2][1] = 22;
	myinfo->pairs[3][0] = 6; myinfo->pairs[3][1] = 20;
	myinfo->pairs[4][0] = 7; myinfo->pairs[4][1] = 18;
	myinfo->pairs[5][0] = 8; myinfo->pairs[5][1] = 16;
	myinfo->pairs[6][0] = 9; myinfo->pairs[6][1] = 14;
	myinfo->pairs[7][0] = 10; myinfo->pairs[7][1] = 12;
	myinfo->pairs[8][0] = 11; myinfo->pairs[8][1] = 10;
	myinfo->pairs[9][0] = 12; myinfo->pairs[9][1] = 8;
	myinfo->pairs[10][0] = 13; myinfo->pairs[10][1] = 6;
	myinfo->pairs[11][0] = 14; myinfo->pairs[11][1] = 4;
	myinfo->pairs[12][0] = 15; myinfo->pairs[12][1] = 2;
	reverse_values(myinfo);
	
	//SPMV_CSRscalar - HST256
	
	myinfo = fill_head(SPMV_CSRscalar, HST256, 10);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 10;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 9;
	myinfo->pairs[2][0] = 4; myinfo->pairs[2][1] = 8;
	myinfo->pairs[3][0] = 5; myinfo->pairs[3][1] = 7;
	myinfo->pairs[4][0] = 7; myinfo->pairs[4][1] = 6;
	myinfo->pairs[5][0] = 8; myinfo->pairs[5][1] = 5;
	myinfo->pairs[6][0] = 10; myinfo->pairs[6][1] = 4;
	myinfo->pairs[7][0] = 11; myinfo->pairs[7][1] = 3;
	myinfo->pairs[8][0] = 12; myinfo->pairs[8][1] = 2;
	myinfo->pairs[9][0] = 14; myinfo->pairs[9][1] = 1;
	reverse_values(myinfo);
	
	////////////////////////////////////////////////////////////////
	
	//Reduction - PF
	
	myinfo = fill_head(Reduction, PF, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 7;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 6;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 5;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 4;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 3;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	save_info = myinfo;
	reverse_values(myinfo);

	//Reduction - GCEDD
	
	myinfo = fill_head(Reduction, GCEDD, 7);
	
	for (int i=0; i<7;i++)
		memcpy(myinfo->pairs[i], save_info->pairs[i], 2 * sizeof(int)); 
	reverse_values(myinfo);

	// Reduction - RCONV 
	
	myinfo = fill_head(Reduction, RCONV, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 26;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 24;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 20;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 16;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 12;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 8;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 4;
	reverse_values(myinfo);
	
	// Reduction -  HST256
	
	myinfo = fill_head(Reduction, HST256, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 9;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 8;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 6;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 5;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 4;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	
	////////////////////////////////////////////////////////////////////////////////
	
	// PF - GCEDD
	
	myinfo = fill_head(PF, GCEDD, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 7;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 6;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 5;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 4;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 3;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	
	// PF - RCONV
	
	myinfo = fill_head(PF, RCONV, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 26;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 24;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 20;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 16;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 12;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 8;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 4;
	reverse_values(myinfo);
	
	// PF - HST256
	
	myinfo = fill_head(PF, HST256, 7);
	
	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 9;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 8;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 6;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 5;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 4;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	
	//////////////////////////////////////////////////////////
	
	// RCONV-GCEDD
	
	myinfo = fill_head(RCONV, GCEDD, 7);
	
	myinfo->pairs[0][0] = 4; myinfo->pairs[0][1] = 7;
	myinfo->pairs[1][0] = 8; myinfo->pairs[1][1] = 6;
	myinfo->pairs[2][0] = 12; myinfo->pairs[2][1] = 5;
	myinfo->pairs[3][0] = 16; myinfo->pairs[3][1] = 4;
	myinfo->pairs[4][0] = 20; myinfo->pairs[4][1] = 3;
	myinfo->pairs[5][0] = 24; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 25; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	
	// RCONV - HST256
	
	myinfo = fill_head(RCONV, HST256, 10);
	
	myinfo->pairs[0][0] = 2; myinfo->pairs[0][1] = 10;
	myinfo->pairs[1][0] = 4; myinfo->pairs[1][1] = 9;
	myinfo->pairs[2][0] = 6; myinfo->pairs[2][1] = 8;
	myinfo->pairs[3][0] = 9; myinfo->pairs[3][1] = 7;
	myinfo->pairs[4][0] = 11; myinfo->pairs[4][1] = 6;
	myinfo->pairs[5][0] = 14; myinfo->pairs[5][1] = 5;
	myinfo->pairs[6][0] = 16; myinfo->pairs[6][1] = 4;
	myinfo->pairs[7][0] = 18; myinfo->pairs[7][1] = 3;
	myinfo->pairs[8][0] = 21; myinfo->pairs[8][1] = 2;
	myinfo->pairs[9][0] = 23; myinfo->pairs[9][1] = 1;
	
	reverse_values(myinfo);

	////////////////////////////////////////////////////////////
		
	// GCEDD - HST256
	
	myinfo = fill_head(GCEDD, HST256, 7);

	myinfo->pairs[0][0] = 1; myinfo->pairs[0][1] = 9;
	myinfo->pairs[1][0] = 2; myinfo->pairs[1][1] = 8;
	myinfo->pairs[2][0] = 3; myinfo->pairs[2][1] = 6;
	myinfo->pairs[3][0] = 4; myinfo->pairs[3][1] = 5;
	myinfo->pairs[4][0] = 5; myinfo->pairs[4][1] = 4;
	myinfo->pairs[5][0] = 6; myinfo->pairs[5][1] = 2;
	myinfo->pairs[6][0] = 7; myinfo->pairs[6][1] = 1;
	reverse_values(myinfo);
	
	return 0;
}

int fill_solo()
{
	
	info_solo[MM].num_configs=8; //max_nm_blocks_per_SM
	info_solo[MM].tpms = (double *)calloc(info_solo[MM].num_configs, sizeof(double));
	
	info_solo[BS].num_configs=8; //max_nm_blocks_per_SM
	info_solo[BS].tpms = (double *)calloc(info_solo[BS].num_configs, sizeof(double));
	
	info_solo[VA].num_configs=8; //max_nm_blocks_per_SM
	info_solo[VA].tpms = (double *)calloc(info_solo[VA].num_configs, sizeof(double));
	
	info_solo[SPMV_CSRscalar].num_configs=16; //max_nm_blocks_per_SM
	info_solo[SPMV_CSRscalar].tpms = (double *)calloc(info_solo[SPMV_CSRscalar].num_configs, sizeof(double));
	
	info_solo[Reduction].num_configs=8; //max_nm_blocks_per_SM
	info_solo[Reduction].tpms = (double *)calloc(info_solo[Reduction].num_configs, sizeof(double));
	
	info_solo[PF].num_configs=8; //max_nm_blocks_per_SM
	info_solo[PF].tpms = (double *)calloc(info_solo[PF].num_configs, sizeof(double));
	
	info_solo[RCONV].num_configs=25; //max_nm_blocks_per_SM
	info_solo[RCONV].tpms = (double *)calloc(info_solo[RCONV].num_configs, sizeof(double));
	
	info_solo[GCEDD].num_configs=8; //max_nm_blocks_per_SM
	info_solo[GCEDD].tpms = (double *)calloc(info_solo[GCEDD].num_configs, sizeof(double));
	
	info_solo[HST256].num_configs=10; //max_nm_blocks_per_SM
	info_solo[HST256].tpms = (double *)calloc(info_solo[HST256].num_configs, sizeof(double));
	
	return 0;
}
