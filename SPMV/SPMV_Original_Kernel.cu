#include "hip/hip_runtime.h"
#include "cudacommon.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include "SPMV.h"
#include <semaphore.h>
#include "../elastic_kernel.h"


texture<float, 1> vecTex;  // vector textures
texture<int2, 1>  vecTexD;


// Texture Readers (used so kernels can be templated)
struct texReaderSP {
   __device__ __forceinline__ float operator()(const int idx) const
   {
       return tex1Dfetch(vecTex, idx);
   }
};

extern t_tqueue *tqueues;


__device__ uint get_smid_SPMV(void) {

     uint ret;

     asm("mov.u32 %0, %smid;" : "=r"(ret) );

     return ret;

}


// ****************************************************************************
// Function: spmv_csr_scalar_kernel
//
// Purpose:
//   Computes sparse matrix - vector multiplication on the GPU using
//   the CSR data storage format, using a thread per row of the sparse
//   matrix; based on Bell (SC09) and Baskaran (IBM Tech Report)
//
// Arguments:
//   val: array holding the non-zero values for the matrix
//   cols: array of column indices for each element of the sparse matrix
//   rowDelimiters: array of size dim+1 holding indices to rows of the matrix
//                  last element is the index one past the last
//                  element of the matrix
//   dim: number of rows in the matrix
//   out: output - result from the spmv calculation
//
// Returns:  nothing
//           out indirectly through a pointer
//
// Programmer: Lukasz Wesolowski
// Creation: June 28, 2010
//
// Modifications:
//
// ****************************************************************************
__global__ void
original_spmv_csr_scalar_kernel(const float * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, float * __restrict__ out)
{
    int myRow = blockIdx.x * blockDim.x + threadIdx.x;
    texReaderSP vecTexReader;

    if (myRow < dim)
    {
        float t = 0.0f;
        int start = rowDelimiters[myRow];
        int end = rowDelimiters[myRow+1];
        for (int j = start; j < end; j++)
        {
            int col = cols[j];
            t += val[j] * vecTexReader(col);
        }
        out[myRow] = t;
    }
}

__global__ void
preemp_SMK_spmv_csr_scalar_kernel(const float * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, float * __restrict__ out,
					   
					   int max_blocks_per_SM, 
						int num_subtask,
						int iter_per_subtask,
						int *cont_SM,
						int *cont_subtask,
						State *status
					   )
{
	int myRow;
	
	__shared__ int s_bid, s_index;
	
	unsigned int SM_id = get_smid_SPMV();
	
	if (threadIdx.x == 0)  
		s_index = atomicAdd(&cont_SM[SM_id],1);
	
	__syncthreads();

	if (s_index > max_blocks_per_SM)
		return;
		
	while (1){
		
		/********** Task Id calculation *************/
		if (threadIdx.x == 0) {
			if (*status == TOEVICT)
				s_bid = -1;
			else
				s_bid = atomicAdd(cont_subtask, 1);
		}
		__syncthreads();
		
		if (s_bid >= num_subtask || s_bid == -1) /* If all subtasks have been executed */
			return;
		
		//for (int iter=0; iter<iter_per_subtask; iter++) {
	
			myRow = s_bid * blockDim.x + threadIdx.x;
			//int myRow = s_bid * blockDim.x * iter_per_subtask + iter * blockDim.x + threadIdx.x;
			texReaderSP vecTexReader;

			if (myRow < dim)
			{
				float t = 0.0f;
				int start = rowDelimiters[myRow];
				int end = rowDelimiters[myRow+1];
				for (int j = start; j < end; j++) {
					int col = cols[j];
					t += val[j] * vecTexReader(col);
				}
				out[myRow] = t;
			}
		//}
	}
}


__global__ void
preemp_SMT_spmv_csr_scalar_kernel(const float * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, float * __restrict__ out,

						int SIMD_min,
						int SIMD_max,
						int num_subtask,
						int iter_per_subtask,
						int *cont_subtask,
						State *status
					   )
{
	int myRow;
	__shared__ int s_bid;
	
	unsigned int SM_id = get_smid_SPMV();
	
	if (SM_id <SIMD_min || SM_id > SIMD_max) /* Only blocks executing within SIMD_min and SIMD_max can progress */ 
			return;
			
	/*if (threadIdx.x == 0 && blockIdx.x % 40 == 0)
		printf("Bloque=%d SM_id=%d\n", blockIdx.x, SM_id);  */
		
	while (1){
		
		/********** Task Id calculation *************/
		if (threadIdx.x == 0) {
			if (*status == TOEVICT) {
				//printf("Sennal eviction %d %d %d %d\n", blockIdx.x, *cont_subtask, iter_per_subtask, num_subtask);
				s_bid = -1;
			}
			else
				s_bid = atomicAdd((int *)cont_subtask, 1);
		}
		
		__syncthreads();
		
		if (s_bid >= num_subtask || s_bid == -1){ /* If all subtasks have been executed */
			//if (blockIdx.x ==0 && threadIdx.x == 0) printf("Blk=%d num_tasks=%d Saliendo por %d\n", blockIdx.x, num_subtask, s_bid );
			return;
		}
		
		//for (int iter=0; iter<iter_per_subtask; iter++) {
	
			myRow = s_bid * blockDim.x + threadIdx.x;
			//int myRow = s_bid * blockDim.x * iter_per_subtask + iter * blockDim.x + threadIdx.x;
			texReaderSP vecTexReader;

			//if (blockIdx.x==0 && threadIdx.x==0)
				//printf("bid=%d Row=%d, start=%d, end=%d ", s_bid, myRow, rowDelimiters[myRow], rowDelimiters[myRow+1]);
			if (myRow < dim)
			{
				float t = 0.0f;
				int start = rowDelimiters[myRow];
				int end = rowDelimiters[myRow+1];
				for (int j = start; j < end; j++) {
					int col = cols[j];
					t += val[j] * vecTexReader(col);
				}
				out[myRow] = t;
				//if (blockIdx.x==0 && threadIdx.x==0)
				//	printf("Result=%f\n", t);
			}
		//}
	}
}

// ****************************************************************************
// Function: spmv_csr_vector_kernel
//
// Purpose:
//   Computes sparse matrix - vector multiplication on the GPU using
//   the CSR data storage format, using a warp per row of the sparse
//   matrix; based on Bell (SC09) and Baskaran (IBM Tech Report)
//
// Arguments:
//   val: array holding the non-zero values for the matrix
//   cols: array of column indices for each element of the sparse matrix
//   rowDelimiters: array of size dim+1 holding indices to rows of the matrix
//                  last element is the index one past the last
//                  element of the matrix
//   dim: number of rows in the matrix
//   out: output - result from the spmv calculation
//
// Returns:  nothing
//           out indirectly through a pointer
//
// Programmer: Lukasz Wesolowski
// Creation: June 28, 2010
//
// Modifications:
//
// ****************************************************************************
template <int BLOCK_SIZE>
__global__ void
original_spmv_csr_vector_kernel(const float * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, float * __restrict__ out)
{
    // Thread ID in block
    int t = threadIdx.x;
    // Thread ID within warp
    int id = t & (warpSize-1);
    int warpsPerBlock = blockDim.x / warpSize;
    // One row per warp
    int myRow = (blockIdx.x * warpsPerBlock) + (t / warpSize);
    // Texture reader for the dense vector
    texReaderSP vecTexReader;

    __shared__ volatile float partialSums[BLOCK_SIZE];

    if (myRow < dim)
    {
        int warpStart = rowDelimiters[myRow];
        int warpEnd = rowDelimiters[myRow+1];
        float mySum = 0;
        for (int j = warpStart + id; j < warpEnd; j += warpSize)
        {
            int col = cols[j];
            mySum += val[j] * vecTexReader(col);
        }
        partialSums[t] = mySum;

        // Reduce partial sums
        if (id < 16) partialSums[t] += partialSums[t+16];
        if (id <  8) partialSums[t] += partialSums[t+ 8];
        if (id <  4) partialSums[t] += partialSums[t+ 4];
        if (id <  2) partialSums[t] += partialSums[t+ 2];
        if (id <  1) partialSums[t] += partialSums[t+ 1];

        // Write result
        if (id == 0)
        {
            out[myRow] = partialSums[t];
        }
    }
}

template <int BLOCK_SIZE>
__global__ void
preemp_SMK_spmv_csr_vector_kernel(const float * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, float * __restrict__ out,
						
						int max_blocks_per_SM, 
						int num_subtask,
						int iter_per_subtask,
						int *cont_SM,
						int *cont_subtask,
						State *status					   
					   )
{
	
	__shared__ int s_bid, s_index;
	__shared__ volatile float partialSums[BLOCK_SIZE];
	
	unsigned int SM_id = get_smid_SPMV();
	
	if (threadIdx.x == 0)  
		s_index = atomicAdd(&cont_SM[SM_id],1);
	
	__syncthreads();

	if (s_index > max_blocks_per_SM)
		return;
		
	while (1){
		
		/********** Task Id calculation *************/
		if (threadIdx.x == 0) {
			if (*status == TOEVICT)
				s_bid = -1;
			else
				s_bid = atomicAdd(cont_subtask, 1);
		}
		__syncthreads();
		
		if (s_bid >= num_subtask || s_bid == -1) /* If all subtasks have been executed */
			return;
		
		for (int iter=0; iter<iter_per_subtask; iter++) {
	
			// Thread ID in block
			int t = threadIdx.x;
			// Thread ID within warp
			int id = t & (warpSize-1);
			int warpsPerBlock = blockDim.x / warpSize;
			// One row per warp
			int myRow = (s_bid * warpsPerBlock * iter_per_subtask + iter * warpsPerBlock) + (t / warpSize);
			// Texture reader for the dense vector
			texReaderSP vecTexReader;

			if (myRow < dim)
			{
				int warpStart = rowDelimiters[myRow];
				int warpEnd = rowDelimiters[myRow+1];
				float mySum = 0;
				for (int j = warpStart + id; j < warpEnd; j += warpSize) {
					int col = cols[j];
					mySum += val[j] * vecTexReader(col);
				}
				partialSums[t] = mySum;

				// Reduce partial sums
				if (id < 16) partialSums[t] += partialSums[t+16];
				if (id <  8) partialSums[t] += partialSums[t+ 8];
				if (id <  4) partialSums[t] += partialSums[t+ 4];
				if (id <  2) partialSums[t] += partialSums[t+ 2];
				if (id <  1) partialSums[t] += partialSums[t+ 1];

				// Write result
				if (id == 0) {
					out[myRow] = partialSums[t];
				}
			}
		}
	}
		
}

template <int BLOCK_SIZE>
__global__ void
preemp_SMT_spmv_csr_vector_kernel(const float * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, float * __restrict__ out,
						
						int SIMD_min,
						int SIMD_max,
						int num_subtask,
						int iter_per_subtask,
						int *cont_subtask,
						State *status				   
					   )
{
	
	__shared__ int s_bid;
	__shared__ volatile float partialSums[BLOCK_SIZE];
	
	unsigned int SM_id = get_smid_SPMV();
	
	if (SM_id <SIMD_min || SM_id > SIMD_max) /* Only blocks executing within SIMD_min and SIMD_max can progress */ 
			return;
		
	while (1){
		
		/********** Task Id calculation *************/
		if (threadIdx.x == 0) {
			if (*status == TOEVICT){
				s_bid = -1;
			}
			else {
				s_bid = atomicAdd(cont_subtask, 1);
			}
		}
		
		__syncthreads();
		
		if (s_bid >= num_subtask || s_bid == -1){ /* If all subtasks have been executed */
			return;
		}
		
		for (int iter=0; iter<iter_per_subtask; iter++) {
	
			// Thread ID in block
			int t = threadIdx.x;
			// Thread ID within warp
			int id = t & (warpSize-1);
			int warpsPerBlock = blockDim.x / warpSize;
			// One row per warp
			int myRow = (s_bid * warpsPerBlock * iter_per_subtask + iter * warpsPerBlock) + (t / warpSize);
			// Texture reader for the dense vector
			texReaderSP vecTexReader;

			if (myRow < dim)
			{
				int warpStart = rowDelimiters[myRow];
				int warpEnd = rowDelimiters[myRow+1];
				float mySum = 0;
				for (int j = warpStart + id; j < warpEnd; j += warpSize) {
					int col = cols[j];
					mySum += val[j] * vecTexReader(col);
				}
				partialSums[t] = mySum;

				// Reduce partial sums
				if (id < 16) partialSums[t] += partialSums[t+16];
				if (id <  8) partialSums[t] += partialSums[t+ 8];
				if (id <  4) partialSums[t] += partialSums[t+ 4];
				if (id <  2) partialSums[t] += partialSums[t+ 2];
				if (id <  1) partialSums[t] += partialSums[t+ 1];

				// Write result
				if (id == 0) {
					out[myRow] = partialSums[t];
				}
			}
		}
	}
		
}



// ****************************************************************************
// Function: spmv_ellpackr_kernel
//
// Purpose:
//   Computes sparse matrix - vector multiplication on the GPU using
//   the ELLPACK-R data storage format; based on Vazquez et al (Univ. of
//   Almeria Tech Report 2009)
//
// Arguments:
//   val: array holding the non-zero values for the matrix in column
//   major format and padded with zeros up to the length of longest row
//   cols: array of column indices for each element of the sparse matrix
//   rowLengths: array storing the length of each row of the sparse matrix
//   dim: number of rows in the matrix
//   out: output - result from the spmv calculation
//
// Returns:  nothing directly
//           out indirectly through a pointer
//
// Programmer: Lukasz Wesolowski
// Creation: June 29, 2010
//
// Modifications:
//
// ****************************************************************************
__global__ void
original_spmv_ellpackr_kernel(const float * __restrict__ val,
                     const int    * __restrict__ cols,
                     const int    * __restrict__ rowLengths,
                     const int dim, float * __restrict__ out)
{
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    texReaderSP vecTexReader;

    if (t < dim)
    {
        float result = 0.0f;
        int max = rowLengths[t];
        for (int i = 0; i < max; i++)
        {
            int ind = i*dim+t;
            result += val[ind] * vecTexReader(cols[ind]);
        }
        out[t] = result;
    }
}


__global__ void
preemp_SMK_spmv_ellpackr_kernel(const float * __restrict__ val,
                     const int    * __restrict__ cols,
                     const int    * __restrict__ rowLengths,
                     const int dim, float * __restrict__ out,
					 						
						int max_blocks_per_SM, 
						int num_subtask,
						int iter_per_subtask,
						int *cont_SM,
						int *cont_subtask,
						State *status					   
					 )
{
	__shared__ int s_bid, s_index;
	
	unsigned int SM_id = get_smid_SPMV();
	
	if (threadIdx.x == 0)  
		s_index = atomicAdd(&cont_SM[SM_id],1);
	
	__syncthreads();

	if (s_index > max_blocks_per_SM)
		return;
		
	while (1){
		
		/********** Task Id calculation *************/
		if (threadIdx.x == 0) {
			if (*status == TOEVICT)
				s_bid = -1;
			else
				s_bid = atomicAdd(cont_subtask, 1);
		}
		__syncthreads();
		
		if (s_bid >= num_subtask || s_bid == -1) /* If all subtasks have been executed */
			return;
		
		for (int iter=0; iter<iter_per_subtask; iter++) {
	
			int t = blockIdx.x * blockDim.x * iter_per_subtask + iter * blockDim.x + threadIdx.x;
			texReaderSP vecTexReader;

			if (t < dim)
			{
				float result = 0.0f;
				int max = rowLengths[t];
				for (int i = 0; i < max; i++) {
					int ind = i*dim+t;
					result += val[ind] * vecTexReader(cols[ind]);
				}
				out[t] = result;
			}
		}
    }
}

__global__ void
preemp_SMT_spmv_ellpackr_kernel(const float * __restrict__ val,
                     const int    * __restrict__ cols,
                     const int    * __restrict__ rowLengths,
                     const int dim, float * __restrict__ out,
					 						
						int SIMD_min,
						int SIMD_max,
						int num_subtask,
						int iter_per_subtask,
						int *cont_subtask,
						State *status			   
					 )
{
	__shared__ int s_bid;
	
	unsigned int SM_id = get_smid_SPMV();
	
	if (SM_id <SIMD_min || SM_id > SIMD_max) /* Only blocks executing within SIMD_min and SIMD_max can progress */ 
			return;
		
	while (1){
		
		/********** Task Id calculation *************/
		if (threadIdx.x == 0) {
			if (*status == TOEVICT)
				s_bid = -1;
			else
				s_bid = atomicAdd(cont_subtask, 1);
		}
		
		__syncthreads();
		
		if (s_bid >= num_subtask || s_bid == -1){ /* If all subtasks have been executed */
			return;
		}
		
		for (int iter=0; iter<iter_per_subtask; iter++) {
	
			int t = blockIdx.x * blockDim.x * iter_per_subtask + iter * blockDim.x + threadIdx.x;
			texReaderSP vecTexReader;

			if (t < dim)
			{
				float result = 0.0f;
				int max = rowLengths[t];
				for (int i = 0; i < max; i++) {
					int ind = i*dim+t;
					result += val[ind] * vecTexReader(cols[ind]);
				}
				out[t] = result;
			}
		}
    }
}


//*************************************************/
// Memory Allocation
//*************************************************/


// float *h_val, *h_vec, *refOut, *h_out;
// int *h_cols, *h_rowDelimiters;

// float *d_val, *d_out, *d_vec;
// int *d_cols, *d_rowDelimiters;

// int numNonZeroes, numRows;
	 
int SPMVcsr_start_kernel(void *arg)
 {
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_SPMV_params * params = (t_SPMV_params *)kstub->params;
	
	// numRows = params->numRows;
    // int nItems = params->nItems;
	//numRows = kstub->kconf.gridsize.x * kstub->kconf.blocksize.x * kstub->kconf.coarsening;

	//Data set 1
	//nItems = numRows * numRows * 0.000005; // 5% of entries will be non-zero

	//Data set 2
	//nItems = numRows * numRows / 14;

	float maxval = 50.0;

	// Allocate and set up host data (only for scalar csr)
	CUDA_SAFE_CALL(hipHostMalloc(&params->h_val, params->nItems * sizeof(float)));
	CUDA_SAFE_CALL(hipHostMalloc(&params->h_vec, params->numRows * sizeof(float)));
	CUDA_SAFE_CALL(hipHostMalloc(&params->h_cols, params->nItems * sizeof(int)));
	CUDA_SAFE_CALL(hipHostMalloc(&params->h_rowDelimiters, (params->numRows + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipHostMalloc(&params->h_out,  params->numRows * sizeof(float)));

	fill(params->h_val, params->nItems, maxval);
	initRandomMatrix_ver3(params->h_cols, params->h_rowDelimiters, params->nItems, params->numRows);
	fill(params->h_vec, params->numRows, maxval);

	// Allocate device memory
	//numNonZeroes = nItems;
	CUDA_SAFE_CALL(hipMalloc(&params->d_val,  params->numNonZeroes * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc(&params->d_cols, params->numNonZeroes * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc(&params->d_vec,  params->numRows * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc(&params->d_out,  params->numRows * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc(&params->d_rowDelimiters, (params->numRows+1) * sizeof(int)));

	// Bind texture for position
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	CUDA_SAFE_CALL(hipBindTexture(0, vecTex, params->d_vec, channelDesc, params->numRows * sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpy(params->d_val, params->h_val,   params->numNonZeroes * sizeof(float),
		hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(params->d_vec, params->h_vec,   params->numRows* sizeof(float),
		hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(params->d_cols, params->h_cols, params->numNonZeroes * sizeof(int),
        hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(params->d_rowDelimiters, params->h_rowDelimiters,
        (params->numRows+1) * sizeof(int), hipMemcpyHostToDevice));
		
	return 0;
 }

int SPMVcsr_start_mallocs(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_SPMV_params * params = (t_SPMV_params *)kstub->params;	
	
	// numRows = params->numRows;
    // int nItems = params->nItems;

	//numRows = kstub->kconf.gridsize.x * kstub->kconf.blocksize.x * kstub->kconf.coarsening;
	//nItems = (int)((double)numRows * (double)(numRows) * 0.000001); // 5% of entries will be non-zero
	//Data set 1
	//nItems = numRows * numRows * 0.000005; // 5% of entries will be non-zero

	//Data set 2
	//nItems = numRows * numRows / 14;

	float maxval = 50.0;

	//printf("Items per row =%d\n", (int)((double)nItems/(double)numRows));

	// Allocate and set up host data (only for scalar csr)
	CUDA_SAFE_CALL(hipHostMalloc(&params->h_val, params->nItems * sizeof(float)));
	CUDA_SAFE_CALL(hipHostMalloc(&params->h_vec, params->numRows * sizeof(float)));
	CUDA_SAFE_CALL(hipHostMalloc(&params->h_cols, params->nItems * sizeof(int)));
	CUDA_SAFE_CALL(hipHostMalloc(&params->h_rowDelimiters, (params->numRows + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipHostMalloc(&params->h_out,  params->numRows * sizeof(float)));

	fill(params->h_val, params->nItems, maxval);
	initRandomMatrix_ver3(params->h_cols, params->h_rowDelimiters, params->nItems, params->numRows);
	fill(params->h_vec, params->numRows, maxval);

	// Allocate device memory
	// numNonZeroes = nItems;
	CUDA_SAFE_CALL(hipMalloc(&params->d_val,  params->numNonZeroes * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc(&params->d_cols, params->numNonZeroes * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc(&params->d_vec,  params->numRows * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc(&params->d_out,  params->numRows * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc(&params->d_rowDelimiters, (params->numRows+1) * sizeof(int)));

	// Bind texture for position
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	CUDA_SAFE_CALL(hipBindTexture(0, vecTex, params->d_vec, channelDesc, params->numRows * sizeof(float)));
	
	return 0;
}

//*************************************************/
// HtD Transfers
//*************************************************/

int SPMVcsr_start_transfers(void *arg){
	
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_SPMV_params * params = (t_SPMV_params *)kstub->params;

	
#if defined(MEMCPY_ASYNC)

	//enqueue_tcomamnd(tqueues, d_val, h_val, numNonZeroes * sizeof(float), hipMemcpyHostToDevice, 
	//					kstub->transfer_s[0], NONBLOCKING, DATA, MEDIUM, kstub);
	hipMemcpyAsync(params->d_val, params->h_val, params->numNonZeroes * sizeof(float), hipMemcpyHostToDevice, kstub->transfer_s[0]);
						
	//enqueue_tcomamnd(tqueues, d_vec, h_vec, numRows * sizeof(float), hipMemcpyHostToDevice, 
	//					kstub->transfer_s[0], NONBLOCKING, DATA, MEDIUM, kstub);
	hipMemcpyAsync(params->d_vec, params->h_vec, params->numRows * sizeof(float), hipMemcpyHostToDevice, kstub->transfer_s[0]);
	
	//enqueue_tcomamnd(tqueues, d_cols, h_cols, numNonZeroes * sizeof(float), hipMemcpyHostToDevice, 
	//					kstub->transfer_s[0], NONBLOCKING, DATA, MEDIUM, kstub);
	hipMemcpyAsync(params->d_cols, params->h_cols, params->numNonZeroes * sizeof(float), hipMemcpyHostToDevice, kstub->transfer_s[0]);

	//enqueue_tcomamnd(tqueues, d_rowDelimiters, h_rowDelimiters, (numRows+1) * sizeof(int), hipMemcpyHostToDevice, 
	//					kstub->transfer_s[0], NONBLOCKING, LAST_TRANSFER, MEDIUM, kstub);
	hipMemcpyAsync(params->d_rowDelimiters, params->h_rowDelimiters, (params->numRows+1) * sizeof(int), hipMemcpyHostToDevice, kstub->transfer_s[0]);
							
#else

	CUDA_SAFE_CALL(hipMemcpy(params->d_val, params->h_val,   params->numNonZeroes * sizeof(float),
              hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(params->d_vec, params->h_vec,   params->numRows* sizeof(float),
              hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(params->d_cols, params->h_cols, params->numNonZeroes * sizeof(int),
              hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(params->d_rowDelimiters, params->h_rowDelimiters,
              (numRows+1) * sizeof(int), hipMemcpyHostToDevice));
#endif

	//kstub->HtD_tranfers_finished = 1;
      
	return 0;
}

//*************************************************/
// DtH transfers and deallocation
//*************************************************/

int SPMVcsr_end_kernel(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_SPMV_params * params = (t_SPMV_params *)kstub->params;

#if defined(MEMCPY_ASYNC)
	//hipEventSynchronize(kstub->end_Exec);

		printf("-->Comienzo de DtH para tarea %d\n", kstub->id);

	//enqueue_tcomamnd(tqueues, h_out, d_out, numRows * sizeof(float), hipMemcpyDeviceToHost, kstub->transfer_s[1] , NONBLOCKING, LAST_TRANSFER, MEDIUM, kstub);
	hipMemcpyAsync(params->h_out, params->d_out, params->numRows * sizeof(float), hipMemcpyDeviceToHost, kstub->transfer_s[1]);
	//hipEventRecord(kstub->end_DtH, kstub->transfer_s[1]);
	
	//kstub->DtH_tranfers_finished = 1;
	
	//	printf("-->Fin de DtH para tarea %d\n", kstub->id);



#else
	
	hipEventSynchronize(kstub->end_Exec);
	
	CUDA_SAFE_CALL(hipMemcpy(params->h_out, params->d_out, params->numRows * sizeof(float),
                  hipMemcpyDeviceToHost));
				 		 
	CUDA_SAFE_CALL(hipFree(params->d_val));
	CUDA_SAFE_CALL(hipFree(params->d_vec));

	CUDA_SAFE_CALL(hipFree(params->d_cols));
	CUDA_SAFE_CALL(hipFree(params->d_rowDelimiters));
	CUDA_SAFE_CALL(hipFree(params->d_out));
#endif
	
	// Compute results on CPU
	params->refOut = new float[params->numRows];
    spmvCpu(params->h_val, params->h_cols, params->h_rowDelimiters, params->h_vec, params->numRows, params->refOut);
	
	if (verifyResults(params->refOut, params->h_out, params->numRows) == false)
		printf("!!!! Error verifying SPMV csr\n");
	
	free(params->refOut);
	hipFree(params->h_vec);
	hipFree(params->h_val);
	hipFree(params->h_cols);
	hipFree(params->h_out);
	hipFree(params->h_rowDelimiters);
	
	return 0;
}
	

int launch_orig_SPMVcsr(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_SPMV_params * params = (t_SPMV_params *)kstub->params;
	
	original_spmv_csr_scalar_kernel<<<kstub->kconf.gridsize.x, kstub->kconf.blocksize.x>>>
			(params->d_val, params->d_cols, params->d_rowDelimiters, params->numRows, params->d_out);
	
	return 0;
}

int launch_preemp_SPMVcsr(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_SPMV_params * params = (t_SPMV_params *)kstub->params;
	
	#ifdef SMT
	preemp_SMT_spmv_csr_scalar_kernel<<<kstub->kconf.numSMs * kstub->kconf.max_persistent_blocks, kstub->kconf.blocksize.x, 0, *(kstub->execution_s)>>>
			(params->d_val, params->d_cols, params->d_rowDelimiters, params->numRows, params->d_out,
			kstub->idSMs[0],
			kstub->idSMs[1],
			kstub->total_tasks, 
			kstub->kconf.coarsening,
			kstub->d_executed_tasks,
			kstub->gm_state
	);
	
	#else
		
	preemp_SMK_spmv_csr_scalar_kernel<<<kstub->kconf.numSMs * kstub->kconf.max_persistent_blocks, kstub->kconf.blocksize.x, 0, *(kstub->execution_s)>>>
			(params->d_val, params->d_cols, params->d_rowDelimiters, params->numRows, params->d_out, 
			kstub->num_blocks_per_SM,
			kstub->total_tasks,
			kstub->kconf.coarsening,
			kstub->d_SMs_cont,
			kstub->d_executed_tasks,
			kstub->gm_state
	);	
		
	#endif
	
	return 0;
}
	 
    
			
	