#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>

// Files
#include <sstream>
#include <string>
#include <fstream>
#include <iostream>

using namespace std;

#include "../elastic_kernel.h"
#include "HST256.h"

#include "../memaddrcnt.cuh"

// uchar *h_Data256;
// uint  *h_HistogramCPU256, *h_HistogramGPU256;
// uchar *d_Data256;
// uint  *d_Histogram256;
// uint byteCount256;

//static const uint PARTIAL_HISTOGRAM256_COUNT = 240;
//static uint *d_PartialHistograms256;

extern t_tqueue *tqueues;

 __device__ uint get_smid_HST256(void) {
	uint ret; 

	asm("mov.u32 %0, %smid;" : "=r"(ret) );

	return ret;
}

inline __device__ void addByte(uint *s_WarpHist, uint data, uint threadTag)
{
    atomicAdd(s_WarpHist + data, 1);
}

inline __device__ void addWord(uint *s_WarpHist, uint data, uint tag)
{
    addByte(s_WarpHist, (data >>  0) & 0xFFU, tag);
    addByte(s_WarpHist, (data >>  8) & 0xFFU, tag);
    addByte(s_WarpHist, (data >> 16) & 0xFFU, tag);
    addByte(s_WarpHist, (data >> 24) & 0xFFU, tag);
}

/**
 * Histogram (CUDA Kernel)
 */
__global__ void
original_histogram256CUDA(uint *d_PartialHistograms256, uint *d_Data256, uint dataCount, 
							int warp_count, int histogram256_threadblock_size, int histogram256_threadblock_memory)
{
    //Per-warp subhistogram storage
    //__shared__ uint s_Hist[histogram256_threadblock_memory];
	extern __shared__ uint s_Hist[];
    uint *s_WarpHist= s_Hist + (threadIdx.x >> LOG2_WARP_SIZE) * HISTOGRAM256_BIN_COUNT;

    //Clear shared memory storage for current threadblock before processing
#pragma unroll

    for (uint i = 0; i < (histogram256_threadblock_memory / histogram256_threadblock_size); i++)
    {
        s_Hist[threadIdx.x + i * histogram256_threadblock_size] = 0;
    }

    //Cycle through the entire data set, update subhistograms for each warp
    const uint tag = threadIdx.x << (UINT_BITS - LOG2_WARP_SIZE);

    __syncthreads();

    for (uint pos = UMAD(blockIdx.x, blockDim.x, threadIdx.x); pos < dataCount; pos += UMUL(blockDim.x, gridDim.x))
    {
        uint data = d_Data256[pos];
        addWord(s_WarpHist, data, tag);
    }

    //Merge per-warp histograms into per-block and write to global memory
    __syncthreads();

    for (uint bin = threadIdx.x; bin < HISTOGRAM256_BIN_COUNT; bin += histogram256_threadblock_size)
    {
        uint sum = 0;

        for (uint i = 0; i < warp_count; i++)
        {
            sum += s_Hist[bin + i * HISTOGRAM256_BIN_COUNT] & TAG_MASK;
        }

        d_PartialHistograms256[blockIdx.x * HISTOGRAM256_BIN_COUNT + bin] = sum;
    }
}

__global__ void
slicing_histogram256CUDA(uint *d_PartialHistograms256, uint *d_Data256, uint dataCount, 
							int warp_count, int histogram256_threadblock_size, int histogram256_threadblock_memory, 
							int gridDimX, int init_block, int *zc_slc)
{
    //Per-warp subhistogram storage
    //__shared__ uint s_Hist[histogram256_threadblock_memory];
	extern __shared__ uint s_Hist[];
	uint *s_WarpHist= s_Hist + (threadIdx.x >> LOG2_WARP_SIZE) * HISTOGRAM256_BIN_COUNT;
	
	if (threadIdx.x == 0) atomicAdd(zc_slc, 1);

    //Clear shared memory storage for current threadblock before processing
#pragma unroll

    for (uint i = 0; i < (histogram256_threadblock_memory / histogram256_threadblock_size); i++)
    {
        s_Hist[threadIdx.x + i * histogram256_threadblock_size] = 0;
    }

    //Cycle through the entire data set, update subhistograms for each warp
    const uint tag = threadIdx.x << (UINT_BITS - LOG2_WARP_SIZE);

    __syncthreads();

    for (uint pos = UMAD(blockIdx.x + init_block, blockDim.x, threadIdx.x); pos < dataCount; pos += UMUL(blockDim.x, gridDimX))
    {
        uint data = d_Data256[pos];
        addWord(s_WarpHist, data, tag);
    }

    //Merge per-warp histograms into per-block and write to global memory
    __syncthreads();

    for (uint bin = threadIdx.x; bin < HISTOGRAM256_BIN_COUNT; bin += histogram256_threadblock_size)
    {
        uint sum = 0;

        for (uint i = 0; i < warp_count; i++)
        {
            sum += s_Hist[bin + i * HISTOGRAM256_BIN_COUNT] & TAG_MASK;
        }

        d_PartialHistograms256[(blockIdx.x + init_block) * HISTOGRAM256_BIN_COUNT + bin] = sum;
    }
}

__global__ void
//__launch_bounds__(192, 8)
profiling_histogram256CUDA(uint *d_PartialHistograms256, uint *d_Data256, uint dataCount, 
							int warp_count, int histogram256_threadblock_size, int histogram256_threadblock_memory,
							int num_subtask,
							int iter_per_subtask,
							int *cont_SM,
							int *cont_subtask,
							State *status)
{
	__shared__ int s_bid, CTA_cont;
	
	unsigned int SM_id = get_smid_HST256();
	
	if (SM_id >= 8){ /* Only blocks executing in first 8 SM  are used for profiling */ 
		//delay();
		return;
	}
	
	if (threadIdx.x == 0) {
		CTA_cont = atomicAdd(&cont_SM[SM_id], 1);
	//	if (SM_id == 7 && CTA_cont == 8)
	//		printf("Aqui\n");
	}
	
	__syncthreads();
	
	if (CTA_cont > SM_id) {/* Only one block makes computation in SM0, two blocks in SM1 and so on */
		//delay();
		return;
	}
	
	//if (threadIdx.x == 0)
	//	printf ("SM=%d CTA = %d\n", SM_id, CTA_cont);

	int cont_task = 0;
	
	while (1){
		
		/********** Task Id calculation *************/
		
		if (threadIdx.x == 0) { 
			if (*status == TOEVICT)
				s_bid = -1;
			else {
				s_bid = atomicAdd(cont_subtask, 1);				//subtask_id
				//printf("Blq=%d cont=%d\n", blockIdx.x, s_bid);
			}
		}
		
		__syncthreads();
		
		//if (s_bid[warpid] >= num_subtask || s_bid[warpid] == -1)
		if (s_bid >=num_subtask || s_bid ==-1) /* If all subtasks have been executed */{
			if (threadIdx.x == 0)
				printf ("SM=%d CTA=%d Executed_tasks= %d \n", SM_id, CTA_cont, cont_task);	 
			return;
		}
		
		if (threadIdx.x == 0) // Acumula numeor de tareas ejecutadas
			 cont_task++;
 
		//Per-warp subhistogram storage
		//__shared__ uint s_Hist[histogram256_threadblock_memory];
		extern __shared__ uint s_Hist[];
		uint *s_WarpHist= s_Hist + (threadIdx.x >> LOG2_WARP_SIZE) * HISTOGRAM256_BIN_COUNT;

		//Clear shared memory storage for current threadblock before processing
	#pragma unroll

		for (uint i = 0; i < (histogram256_threadblock_memory / histogram256_threadblock_size); i++)
		{
			s_Hist[threadIdx.x + i * histogram256_threadblock_size] = 0;
		}

		//Cycle through the entire data set, update subhistograms for each warp
		const uint tag = threadIdx.x << (UINT_BITS - LOG2_WARP_SIZE);

		__syncthreads();

		for (uint pos = UMAD(s_bid, blockDim.x, threadIdx.x); pos < dataCount; pos += UMUL(blockDim.x, gridDim.x))
		{
			uint data = d_Data256[pos];
			addWord(s_WarpHist, data, tag);
		}

		//Merge per-warp histograms into per-block and write to global memory
		__syncthreads();

		for (uint bin = threadIdx.x; bin < HISTOGRAM256_BIN_COUNT; bin += histogram256_threadblock_size)
		{
			uint sum = 0;

			for (uint i = 0; i < warp_count; i++)
			{
				sum += s_Hist[bin + i * HISTOGRAM256_BIN_COUNT] & TAG_MASK;
			}

			d_PartialHistograms256[s_bid * HISTOGRAM256_BIN_COUNT + bin] = sum;
		}
	}
}


__global__ void
//__launch_bounds__(192, 8)
SMT_histogram256CUDA_kk(uint *d_PartialHistograms256, uint *d_Data256, uint dataCount,
						int warp_count, int histogram256_threadblock_size, int histogram256_threadblock_memory,
						int SIMD_min, int SIMD_max,
						int num_subtask, int iter_per_subtask, int *cont_subtask, State *status)
{
	__shared__ int s_bid;
	
	unsigned int SM_id = get_smid_HST256();
	
	if (SM_id <SIMD_min || SM_id > SIMD_max) /* Only blocks executing within SIMD_min and SIMD_max can progress */ 
			return;
	
	while (1){
		
		/********** Task Id calculation *************/
		
		if (threadIdx.x == 0) { 
			if (*status == TOEVICT)
				s_bid = -1;
			else {
				s_bid = atomicAdd(cont_subtask, 1);				//subtask_id
				//printf("Blq=%d cont=%d\n", blockIdx.x, s_bid);
			}
		}
		
		__syncthreads();
		
		//if (s_bid[warpid] >= num_subtask || s_bid[warpid] == -1)
		if (s_bid >=num_subtask || s_bid ==-1) /* If all subtasks have been executed */{
			//if (threadIdx.x == 0)  printf("El bloque %d se sale con %d\n", blockIdx.x, s_bid); 
			return;
		}

		//Per-warp subhistogram storage
		//__shared__ uint s_Hist[histogram256_threadblock_memory];
		extern __shared__ uint s_Hist[];
		uint *s_WarpHist= s_Hist + (threadIdx.x >> LOG2_WARP_SIZE) * HISTOGRAM256_BIN_COUNT;

		//Clear shared memory storage for current threadblock before processing
	#pragma unroll

		for (uint i = 0; i < (histogram256_threadblock_memory / histogram256_threadblock_size); i++)
		{
			s_Hist[threadIdx.x + i * histogram256_threadblock_size] = 0;
		}

		//Cycle through the entire data set, update subhistograms for each warp
		const uint tag = threadIdx.x << (UINT_BITS - LOG2_WARP_SIZE);

		__syncthreads();

		for (uint pos = UMAD(s_bid, blockDim.x, threadIdx.x); pos < dataCount; pos += UMUL(blockDim.x, num_subtask))
		{
			uint data = d_Data256[pos];
			addWord(s_WarpHist, data, tag);
		}

		//Merge per-warp histograms into per-block and write to global memory
		__syncthreads();

		for (uint bin = threadIdx.x; bin < HISTOGRAM256_BIN_COUNT; bin += histogram256_threadblock_size)
		{
			uint sum = 0;

			for (uint i = 0; i < warp_count; i++)
			{
				sum += s_Hist[bin + i * HISTOGRAM256_BIN_COUNT] & TAG_MASK;
			}

			d_PartialHistograms256[s_bid * HISTOGRAM256_BIN_COUNT + bin] = sum;
		}
	}
}

__global__ void
//__launch_bounds__(192, 8)
SMT_histogram256CUDA(uint *d_PartialHistograms256, uint *d_Data256, uint dataCount,
					int warp_count, int histogram256_threadblock_size, int histogram256_threadblock_memory,
					int SIMD_min, int SIMD_max,
					int num_subtask, int iter_per_subtask, int *cont_subtask, State *status, int tasks)
{
	__shared__ int s_bid;
	
	unsigned int SM_id = get_smid_HST256();
	
	if (SM_id <SIMD_min || SM_id > SIMD_max) /* Only blocks executing within SIMD_min and SIMD_max can progress */ 
			return;

	//Per-warp subhistogram storage
	//__shared__ uint s_Hist[histogram256_threadblock_memory];
	extern __shared__ uint s_Hist[];
	uint *s_WarpHist, tag, pos;
	uint sum;
			
	while (1){
		
		/********** Task Id calculation *************/
		
		if (threadIdx.x == 0) { 
			if (*status == TOEVICT)
				s_bid = -1;
			else {
				s_bid = atomicAdd(cont_subtask, 1);				//subtask_id
				//printf("Blq=%d cont=%d\n", blockIdx.x, s_bid);
			}
		}
		
		__syncthreads();
		
		//if (s_bid[warpid] >= num_subtask || s_bid[warpid] == -1)
		if (s_bid >=num_subtask || s_bid ==-1) /* If all subtasks have been executed */{				
			return;
		}

		s_WarpHist= s_Hist + (threadIdx.x >> LOG2_WARP_SIZE) * HISTOGRAM256_BIN_COUNT;

		//Clear shared memory storage for current threadblock before processing
		#pragma unroll

		for (uint i = 0; i < (histogram256_threadblock_memory / histogram256_threadblock_size); i++)
		{
			s_Hist[threadIdx.x + i * histogram256_threadblock_size] = 0;
		}

		//Cycle through the entire data set, update subhistograms for each warp
		tag = threadIdx.x << (UINT_BITS - LOG2_WARP_SIZE);

		__syncthreads();
		
		if(s_bid < tasks){
			pos = UMAD(s_bid, blockDim.x, threadIdx.x);
		}else{
			pos = UMAD(s_bid % tasks, blockDim.x, threadIdx.x) + (s_bid / tasks) * iter_per_subtask * UMUL(blockDim.x, tasks);
		}
		
		for(int iter = 0; iter < iter_per_subtask; iter++){
			if(pos < dataCount){
				uint data = d_Data256[pos];
				addWord(s_WarpHist, data, tag);
			}
			
			pos += UMUL(blockDim.x, tasks);
		}

		//Merge per-warp histograms into per-block and write to global memory
		__syncthreads();

		for (uint bin = threadIdx.x; bin < HISTOGRAM256_BIN_COUNT; bin += histogram256_threadblock_size)
		{
			if(s_bid < tasks){
				//d_PartialHistograms256[(s_bid % tasks) * HISTOGRAM256_BIN_COUNT + bin] = 0;
				d_PartialHistograms256[blockIdx.x * HISTOGRAM256_BIN_COUNT + bin] = 0;
			}
			
			sum = 0;
			
			for (uint i = 0; i < warp_count; i++)
			{
				sum += s_Hist[bin + i * HISTOGRAM256_BIN_COUNT] & TAG_MASK;
			}

			//d_PartialHistograms256[(s_bid % tasks) * HISTOGRAM256_BIN_COUNT + bin] += sum;
			d_PartialHistograms256[blockIdx.x * HISTOGRAM256_BIN_COUNT + bin] += sum;
		}
	}
}

__global__ void
//__launch_bounds__(192, 8)
memaddr_SMT_histogram256CUDA(uint *d_PartialHistograms256, uint *d_Data256, uint dataCount,
					int warp_count, int histogram256_threadblock_size, int histogram256_threadblock_memory,
					int *numUniqueAddr, int SIMD_min, int SIMD_max,
					int num_subtask, int iter_per_subtask, int *cont_subtask, State *status, int tasks)
{
	__shared__ int s_bid;
	
	unsigned int SM_id = get_smid_HST256();
	
	if (SM_id <SIMD_min || SM_id > SIMD_max) /* Only blocks executing within SIMD_min and SIMD_max can progress */ 
			return;

	//Per-warp subhistogram storage
	//__shared__ uint s_Hist[histogram256_threadblock_memory];
	extern __shared__ uint s_Hist[];
	uint *s_WarpHist, tag, pos;
	uint sum;
			
	while (1){
		
		/********** Task Id calculation *************/
		
		if (threadIdx.x == 0) { 
			if (*status == TOEVICT)
				s_bid = -1;
			else {
				s_bid = atomicAdd(cont_subtask, 1);				//subtask_id
				//printf("Blq=%d cont=%d\n", blockIdx.x, s_bid);
			}
		}
		
		__syncthreads();
		
		//if (s_bid[warpid] >= num_subtask || s_bid[warpid] == -1)
		if (s_bid >=num_subtask || s_bid ==-1) /* If all subtasks have been executed */{				
			return;
		}

		s_WarpHist= s_Hist + (threadIdx.x >> LOG2_WARP_SIZE) * HISTOGRAM256_BIN_COUNT;

		//Clear shared memory storage for current threadblock before processing
		#pragma unroll

		for (uint i = 0; i < (histogram256_threadblock_memory / histogram256_threadblock_size); i++)
		{
			if ( s_bid == 0 )
				get_conflicting_banks( (intptr_t) &s_Hist[threadIdx.x + i * histogram256_threadblock_size], &numUniqueAddr[1] );				
			s_Hist[threadIdx.x + i * histogram256_threadblock_size] = 0;
		}

		//Cycle through the entire data set, update subhistograms for each warp
		tag = threadIdx.x << (UINT_BITS - LOG2_WARP_SIZE);

		__syncthreads();
		
		if(s_bid < tasks){
			pos = UMAD(s_bid, blockDim.x, threadIdx.x);
		}else{
			pos = UMAD(s_bid % tasks, blockDim.x, threadIdx.x) + (s_bid / tasks) * iter_per_subtask * UMUL(blockDim.x, tasks);
		}
		
		for(int iter = 0; iter < iter_per_subtask; iter++){
			if(pos < dataCount){
#if defined(COUNT_ALL_TASKS)
				if ( s_bid == 0 )
#endif
				{
					get_unique_lines((intptr_t) &d_Data256[pos], numUniqueAddr);
				}
				uint data = d_Data256[pos];
#if defined(COUNT_ALL_TASKS)
				if ( s_bid == 0 )
#endif
				{
					get_conflicting_banks( (intptr_t) s_WarpHist+data, &numUniqueAddr[1] );
				}
				addWord(s_WarpHist, data, tag);
			}
			
			pos += UMUL(blockDim.x, tasks);
		}

		//Merge per-warp histograms into per-block and write to global memory
		__syncthreads();

		for (uint bin = threadIdx.x; bin < HISTOGRAM256_BIN_COUNT; bin += histogram256_threadblock_size)
		{
			if(s_bid < tasks){
				//d_PartialHistograms256[(s_bid % tasks) * HISTOGRAM256_BIN_COUNT + bin] = 0;
#if defined(COUNT_ALL_TASKS)
				if ( s_bid == 0 )
#endif
				{
					get_unique_lines((intptr_t) &d_PartialHistograms256[blockIdx.x * HISTOGRAM256_BIN_COUNT + bin], numUniqueAddr);
				}
				d_PartialHistograms256[blockIdx.x * HISTOGRAM256_BIN_COUNT + bin] = 0;
			}
			
			sum = 0;
			
			for (uint i = 0; i < warp_count; i++)
			{
#if defined(COUNT_ALL_TASKS)
				if ( s_bid == 0 )
#endif
				{
					get_conflicting_banks( (intptr_t) &s_Hist[bin + i * HISTOGRAM256_BIN_COUNT], &numUniqueAddr[1] );
				}
				sum += s_Hist[bin + i * HISTOGRAM256_BIN_COUNT] & TAG_MASK;
			}

			//d_PartialHistograms256[(s_bid % tasks) * HISTOGRAM256_BIN_COUNT + bin] += sum;
#if defined(COUNT_ALL_TASKS)
				if ( s_bid == 0 )
#endif
			{
				get_unique_lines((intptr_t) &d_PartialHistograms256[(s_bid % tasks) * HISTOGRAM256_BIN_COUNT + bin], numUniqueAddr);
			}
			d_PartialHistograms256[blockIdx.x * HISTOGRAM256_BIN_COUNT + bin] += sum;
		}
	}
}

__global__ void
//__launch_bounds__(192, 8)
SMK_histogram256CUDA(uint *d_PartialHistograms256, uint *d_Data256, uint dataCount,
					int warp_count, int histogram256_threadblock_size, int histogram256_threadblock_memory,
					int max_blocks_per_SM,
					int gridDimX, 
					int num_subtask,
					int iter_per_subtask,
					int *cont_SM,
					int *cont_subtask,
					State *status,
					int tasks
)
{
	__shared__ int s_bid, s_index;
	
	unsigned int SM_id = get_smid_HST256();
	
	if (threadIdx.x == 0)  
		s_index = atomicAdd(&cont_SM[SM_id],1);
	
	__syncthreads();

	if (s_index > max_blocks_per_SM)
		return;
	
	uint sum;
	
	while (1){
		
		/********** Task Id calculation *************/
		if (threadIdx.x == 0) {
			if (*status == TOEVICT)
				s_bid = -1;
			else
				s_bid = atomicAdd(cont_subtask, 1);
		}
		
		__syncthreads();
		
		if (s_bid >= num_subtask || s_bid == -1) /* If all subtasks have been executed */
			return;
		
		//Per-warp subhistogram storage
		//__shared__ uint s_Hist[histogram256_threadblock_memory];
		extern __shared__ uint s_Hist[];
		uint *s_WarpHist= s_Hist + (threadIdx.x >> LOG2_WARP_SIZE) * HISTOGRAM256_BIN_COUNT;

		//Clear shared memory storage for current threadblock before processing
	#pragma unroll

		for (uint i = 0; i < (histogram256_threadblock_memory / histogram256_threadblock_size); i++)
		{
			s_Hist[threadIdx.x + i * histogram256_threadblock_size] = 0;
		}

		//Cycle through the entire data set, update subhistograms for each warp
		const uint tag = threadIdx.x << (UINT_BITS - LOG2_WARP_SIZE);

		__syncthreads();

		//for (uint pos = UMAD(s_bid, blockDim.x, threadIdx.x); pos < dataCount; pos += UMUL(blockDim.x, gridDim.x))
		//{
		//	uint data = d_Data256[pos];
		//	addWord(s_WarpHist, data, tag);
		//}

		for (uint pos = UMAD(s_bid, blockDim.x, threadIdx.x); pos < dataCount; pos += UMUL(blockDim.x, num_subtask))
		{
			uint data = d_Data256[pos];
			addWord(s_WarpHist, data, tag);
		}
		

		//Merge per-warp histograms into per-block and write to global memory
		__syncthreads();

		for (uint bin = threadIdx.x; bin < HISTOGRAM256_BIN_COUNT; bin += histogram256_threadblock_size)
		{
			if(s_bid < tasks){
				//d_PartialHistograms256[(s_bid % tasks) * HISTOGRAM256_BIN_COUNT + bin] = 0;
				d_PartialHistograms256[blockIdx.x * HISTOGRAM256_BIN_COUNT + bin] = 0;
			}
			
			sum = 0;
			
			for (uint i = 0; i < warp_count; i++)
			{
				sum += s_Hist[bin + i * HISTOGRAM256_BIN_COUNT] & TAG_MASK;
			}

			//d_PartialHistograms256[(s_bid % tasks) * HISTOGRAM256_BIN_COUNT + bin] += sum;
			d_PartialHistograms256[blockIdx.x * HISTOGRAM256_BIN_COUNT + bin] += sum;
		}
	}
}

int HST256_start_kernel(void *arg) 
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_HST256_params * params = (t_HST256_params *)kstub->params;
	
	//byteCount256 = params->byteCount256;
	//Data set 1 
	//byteCount256 = 64 * 1048576 * 6;
	
	//Data set 2
	//byteCount256 = 64 * 1048576 * 6 * 6;

	// h_Data256         = (uchar *)malloc(byteCount256);
    // h_HistogramCPU256 = (uint *)malloc(HISTOGRAM256_BIN_COUNT * sizeof(uint));
    // h_HistogramGPU256 = (uint *)malloc(HISTOGRAM256_BIN_COUNT * sizeof(uint));
	
	hipHostMalloc(&params->h_Data256, params->byteCount256);
	hipHostMalloc(&params->h_HistogramGPU256, HISTOGRAM256_BIN_COUNT * sizeof(uint));
	hipHostMalloc(&params->h_PartialHistograms256, PARTIAL_HISTOGRAM256_COUNT * HISTOGRAM256_BIN_COUNT * sizeof(uint));
	
	srand(2009);

    for (uint i = 0; i < params->byteCount256; i++)
    {
        params->h_Data256[i] = rand() % 256;
    }
	
	checkCudaErrors(hipMalloc((void **)&params->d_Data256, params->byteCount256));
    checkCudaErrors(hipMalloc((void **)&params->d_Histogram256, HISTOGRAM256_BIN_COUNT * sizeof(uint)));
    checkCudaErrors(hipMemcpy(params->d_Data256, params->h_Data256, params->byteCount256, hipMemcpyHostToDevice));
	
	checkCudaErrors(hipMalloc((void **)&params->d_PartialHistograms256, PARTIAL_HISTOGRAM256_COUNT * HISTOGRAM256_BIN_COUNT * sizeof(uint)));
	checkCudaErrors(hipMemset(params->d_PartialHistograms256, 0, PARTIAL_HISTOGRAM256_COUNT * HISTOGRAM256_BIN_COUNT * sizeof(uint)));
	
	return 0;
}

int HST256_start_mallocs(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_HST256_params * params = (t_HST256_params *)kstub->params;

	// globalmemory position for launched ctas counter
	hipMalloc((void **)&params->zc_slc, sizeof(int));
	
	//byteCount256 = params->byteCount256;
	
	//Data set 1 
	//byteCount256 = 64 * 1048576 * 6;
	
	//Data set 2
	//byteCount256 = 64 * 1048576 * 6 * 6;
	
#if defined(MEMCPY_SYNC) || defined(MEMCPY_ASYNC)
	hipHostMalloc(&params->h_Data256, params->byteCount256);
	hipHostMalloc(&params->h_HistogramGPU256, HISTOGRAM256_BIN_COUNT * sizeof(uint));
	hipHostMalloc(&params->h_PartialHistograms256, PARTIAL_HISTOGRAM256_COUNT * HISTOGRAM256_BIN_COUNT * sizeof(uint));

	checkCudaErrors(hipMalloc((void **)&params->d_Data256, params->byteCount256));
    checkCudaErrors(hipMalloc((void **)&params->d_Histogram256, HISTOGRAM256_BIN_COUNT * sizeof(uint)));	
	checkCudaErrors(hipMalloc((void **)&params->d_PartialHistograms256, PARTIAL_HISTOGRAM256_COUNT * HISTOGRAM256_BIN_COUNT * sizeof(uint)));
#else
	#ifdef MANAGED_MEM

	hipMallocManaged(&params->h_Data256, params->byteCount256);
	hipMallocManaged(&params->h_HistogramCPU256, HISTOGRAM256_BIN_COUNT * sizeof(uint));
	hipMallocManaged(&params->h_HistogramGPU256, HISTOGRAM256_BIN_COUNT * sizeof(uint));

	#else
		printf("No transfer model: Exiting ...\n");
		exit(-1);
	#endif
#endif

	// Verify that allocations succeeded
    if (params->h_Data256 == NULL || params->h_HistogramGPU256 == NULL || params->h_PartialHistograms256 == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    srand(2009);

    for (uint i = 0; i < params->byteCount256; i++)
    {
        params->h_Data256[i] = rand() % 256;
    }
	
	checkCudaErrors(hipMemset(params->d_PartialHistograms256, 0, PARTIAL_HISTOGRAM256_COUNT * HISTOGRAM256_BIN_COUNT * sizeof(uint)));

	return 0;
}

int HST256_start_transfers(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_HST256_params * params = (t_HST256_params *)kstub->params;
	
	//byteCount256 = params->byteCount256;
	
	//Data set 1 
	//byteCount256 = 64 * 1048576 * 6;
	
	//Data set 2
	//byteCount256 = 64 * 1048576 * 6 * 6;
	
#ifdef MEMCPY_SYNC
	enqueue_tcomamnd(tqueues, params->d_Data256, params->h_Data256, params->byteCount256, hipMemcpyHostToDevice, 0, BLOCKING, DATA, LOW, kstub);
	kstub->HtD_tranfers_finished = 1;

	
#else
	
	#ifdef MEMCPY_ASYNC
	
	//enqueue_tcomamnd(tqueues, d_Data256, h_Data256, byteCount256, hipMemcpyHostToDevice, 0, NONBLOCKING, LAST_TRANSFER, MEDIUM, kstub);
	hipMemcpyAsync(params->d_Data256, params->h_Data256, params->byteCount256, hipMemcpyHostToDevice, kstub->transfer_s[0]);

	#else
	#ifdef MANAGED_MEM

	hipDeviceProp_t p;
    hipGetDeviceProperties(&p, kstub->deviceId);
	
	if (p.concurrentManagedAccess)
	{
		err = hipMemPrefetchAsync(params->h_Data256, params->byteCount256, kstub->deviceId);
		if ( err != hipSuccess) {
			printf("Error in vAdd:hipMemPrefetchAsync\n");
			exit(EXIT_FAILURE);
		}
		err = hipMemPrefetchAsync(params->h_HistogramCPU256, HISTOGRAM256_BIN_COUNT * sizeof(uint), kstub->deviceId);
		if ( err != hipSuccess) {
			printf("Error in vAdd:hipMemPrefetchAsync\n");
			exit(EXIT_FAILURE);
		}
		err = hipMemPrefetchAsync(params->h_HistogramGPU256, HISTOGRAM256_BIN_COUNT * sizeof(uint), kstub->deviceId);
		if ( err != hipSuccess) {
			printf("Error in vAdd:hipMemPrefetchAsync\n");
			exit(EXIT_FAILURE);
		}
	}
	//hipEventRecord(kstub->end_HtD, kstub->transfer_s[0]);
	
	//hipStreamSynchronize(kstub->transfer_s[0]);
	kstub->HtD_tranfers_finished = 1;

	#endif
	#endif
#endif

	return 0;
}

int HST256_end_kernel(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_HST256_params * params = (t_HST256_params *)kstub->params;
	
/*#ifdef MEMCPY_SYNC

	hipEventSynchronize(kstub->end_Exec);

	enqueue_tcomamnd(tqueues, params->h_HistogramGPU256, params->d_Histogram256, HISTOGRAM256_BIN_COUNT * sizeof(uint), hipMemcpyDeviceToHost, 0, BLOCKING, DATA, LOW, kstub);
	 
#else
	#ifdef MEMCPY_ASYNC
	//enqueue_tcomamnd(tqueues, h_HistogramGPU256, d_Histogram256, HISTOGRAM256_BIN_COUNT * sizeof(uint), hipMemcpyDeviceToHost, kstub->transfer_s[1] , NONBLOCKING, LAST_TRANSFER, MEDIUM, kstub);
	hipMemcpyAsync(params->h_HistogramGPU256, params->d_PartialHistograms256, HISTOGRAM256_BIN_COUNT * sizeof(uint), hipMemcpyDeviceToHost, kstub->transfer_s[1]);
	
	#else
		#ifdef MANAGED_MEM
			hipStreamSynchronize(*(kstub->execution_s)); // To be sure kernel execution has finished before processing output data
		#endif
	#endif
#endif*/

	checkCudaErrors(hipMemcpy(params->h_PartialHistograms256, params->d_PartialHistograms256, PARTIAL_HISTOGRAM256_COUNT * HISTOGRAM256_BIN_COUNT * sizeof(uint), hipMemcpyDeviceToHost));
		
	hipDeviceSynchronize();
		
//	for(int iteracion = 0; iteracion < PARTIAL_HISTOGRAM256_COUNT * HISTOGRAM256_BIN_COUNT; iteracion++)
//		printf("%u ", params->h_PartialHistograms256[iteracion]);
//	printf("\n");

	return 0;
}

/*int HST256_end_kernel_dummy(void *arg)
{	
	checkCudaErrors(hipMemcpy(h_HistogramGPU256, d_Histogram256, HISTOGRAM256_BIN_COUNT * sizeof(uint), hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(d_PartialHistograms256));
	checkCudaErrors(hipFree(d_Histogram256));
    checkCudaErrors(hipFree(d_Data256));
    // free(h_HistogramGPU256);
    // free(h_HistogramCPU256);
    // free(h_Data256);
	
	hipHostFree(h_HistogramGPU256);
    hipHostFree(h_HistogramCPU256);
    hipHostFree(h_Data256);
	
    return 0;
}*/
 
int launch_orig_HST256(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_HST256_params * params = (t_HST256_params *)kstub->params;
	
	//for ( int i = 0; i < kstub->kconf.coarsening; i++ )
		original_histogram256CUDA<<<kstub->kconf.gridsize.x, kstub->kconf.blocksize.x, params->histogram256_threadblock_memory * sizeof(uint)>>>(
			params->d_PartialHistograms256,
			(uint *)params->d_Data256,
			params->byteCount256 / sizeof(uint),
			
			params->warp_count,
			params->histogram256_threadblock_size,
			params->histogram256_threadblock_memory
    	);

	return 0;
}

int launch_slc_HST256(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_HST256_params * params = (t_HST256_params *)kstub->params;
	
	slicing_histogram256CUDA<<<kstub->total_tasks, kstub->kconf.blocksize.x, params->histogram256_threadblock_memory * sizeof(uint), *(kstub->execution_s)>>>(
        params->d_PartialHistograms256,
        (uint *)params->d_Data256,
        params->byteCount256 / sizeof(uint),
		
		params->warp_count,
		params->histogram256_threadblock_size,
		params->histogram256_threadblock_memory,
		params->gridDimX,
		kstub->kconf.initial_blockID,
		params->zc_slc
    );

	return 0;
}

int prof_HST256(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_HST256_params * params = (t_HST256_params *)kstub->params;
	
	profiling_histogram256CUDA<<<kstub->kconf.numSMs * kstub->kconf.max_persistent_blocks, kstub->kconf.blocksize.x, params->histogram256_threadblock_memory * sizeof(uint), *(kstub->execution_s)>>>(
			params->d_PartialHistograms256,
			(uint *)params->d_Data256,
			params->byteCount256 / sizeof(uint),
			
			params->warp_count,
			params->histogram256_threadblock_size,
			params->histogram256_threadblock_memory,
			
			kstub->total_tasks,
			kstub->kconf.coarsening,
			kstub->d_SMs_cont,
			kstub->d_executed_tasks,
			&kstub->gm_state[kstub->stream_index]);
	return 0;
}

int launch_preemp_HST256(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_HST256_params * params = (t_HST256_params *)kstub->params;
	
	//Numero de totales a ejecutar
	//int tasks = params->byteCount256 / (sizeof(uint) * kstub->kconf.blocksize.x * kstub->kconf.coarsening);
	
	#ifdef SMT
		if ( !(kstub->memaddr_profile) )	
			SMT_histogram256CUDA<<<kstub->kconf.numSMs * kstub->kconf.max_persistent_blocks, kstub->kconf.blocksize.x, params->histogram256_threadblock_memory * sizeof(uint), *(kstub->execution_s)>>>(
				params->d_PartialHistograms256,
				(uint *)params->d_Data256,
				params->byteCount256 / sizeof(uint),
			
				params->warp_count,
				params->histogram256_threadblock_size,
				params->histogram256_threadblock_memory,
			
				kstub->idSMs[0],
				kstub->idSMs[1],
				kstub->total_tasks,
				kstub->kconf.coarsening,
				kstub->d_executed_tasks,
				&(kstub->gm_state[kstub->stream_index]),
				kstub->kconf.gridsize.x);
		else
			memaddr_SMT_histogram256CUDA<<<kstub->kconf.numSMs * kstub->kconf.max_persistent_blocks, kstub->kconf.blocksize.x, params->histogram256_threadblock_memory * sizeof(uint), *(kstub->execution_s)>>>(
				params->d_PartialHistograms256,
				(uint *)params->d_Data256,
				params->byteCount256 / sizeof(uint),
			
				params->warp_count,
				params->histogram256_threadblock_size,
				params->histogram256_threadblock_memory,
			
				kstub->d_numUniqueAddr,		
				kstub->idSMs[0],
				kstub->idSMs[1],
				kstub->total_tasks,
				kstub->kconf.coarsening,
				kstub->d_executed_tasks,
				&(kstub->gm_state[kstub->stream_index]),
				kstub->kconf.gridsize.x);
		
		// SMT_histogram256CUDA_kk<<<kstub->kconf.numSMs * kstub->kconf.max_persistent_blocks, kstub->kconf.blocksize.x, params->histogram256_threadblock_memory * sizeof(uint), *(kstub->execution_s)>>>(
			// params->d_PartialHistograms256,
			// (uint *)params->d_Data256,
			// params->byteCount256 / sizeof(uint),
			
			// params->warp_count,
			// params->histogram256_threadblock_size,
			// params->histogram256_threadblock_memory,
			
			// kstub->idSMs[0],
			// kstub->idSMs[1],
			// kstub->total_tasks,
			// kstub->kconf.coarsening,
			// kstub->d_executed_tasks,
			// &(kstub->gm_state[kstub->stream_index])
		// );
	#else
		SMK_histogram256CUDA<<<kstub->kconf.numSMs * kstub->kconf.max_persistent_blocks, kstub->kconf.blocksize.x, params->histogram256_threadblock_memory * sizeof(uint), *(kstub->execution_s)>>>(
			params->d_PartialHistograms256,
			(uint *)params->d_Data256,
			params->byteCount256 / sizeof(uint),
			
			params->warp_count,
			params->histogram256_threadblock_size,
			params->histogram256_threadblock_memory,
			
			kstub->num_blocks_per_SM,
			kstub->total_tasks,
			kstub->kconf.coarsening,
			kstub->d_SMs_cont,
			kstub->d_executed_tasks,
			&(kstub->gm_state[kstub->stream_index]),
			kstub->kconf.gridsize.x
		);
	#endif
	
	return 0;
}
