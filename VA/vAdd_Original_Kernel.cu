#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */
 
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_") 
#include <hip/hip_runtime.h>
#include <helper_functions.h>   // helper functions for string parsing
#include <hip/hip_runtime_api.h>        // helper functions CUDA error checking and initialization
#include <semaphore.h>
#include "../elastic_kernel.h"
#include "VA.h"
#include "../memaddrcnt.cuh"

//extern t_tqueue *tqueues;

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */

 
  __device__ uint get_smid_VA(void) {

     uint ret;

     asm("mov.u32 %0, %smid;" : "=r"(ret) );

     return ret;

}

__global__ void
original_vectorAdd(const float *A, const float *B, float *C, int iter_per_block, int numelements)
{
	
	for (int k=0; k<iter_per_block; k++) {
			
		//const int i = k * gridDim.x * blockDim.x + j * gridDim.x * blockDim.x +  
		//blockIdx.x * blockDim.x + threadIdx.x;
			
		const int i = blockIdx.x * blockDim.x * iter_per_block + k * blockDim.x + threadIdx.x;
			
		if (i < numelements)
				C[i] = A[i] + B[i];
	}
	

}

__global__ void
preempt_SMK_vectorAdd(const float *A, const float *B, float *C, int numelements, 
			int max_blocks_per_SM, 
			int num_subtask, int iter_per_subtask, int *cont_SM, int *cont_subtask, State *status)
{
	int i;
	
	__shared__ int s_bid, s_index;
	
	unsigned int SM_id = get_smid_VA();
	
	if (threadIdx.x == 0)  
		s_index = atomicAdd(&cont_SM[SM_id],1);
	
	__syncthreads();

	if (s_index > max_blocks_per_SM)
		return;
	
	//unsigned int SM_id = get_smid_VA();
	
	//int warpid = threadIdx.x >> 5;
	
	//int thIdxwarp = threadIdx.x & 0x1F;
	
	//if (SM_id <SIMD_min || SM_id > SIMD_max) /* Only blocks executing within SIMD_min and SIMD_max can progress */ 
	//		return;
		
	while (1){
		
		/********** Task Id calculation *************/
		
		if (threadIdx.x == 0) {
			if (*status == TOEVICT)
				s_bid = -1;
			else
				s_bid = atomicAdd(cont_subtask, 1); //subtask_id
		}
		
		//if ( thIdxwarp == 0) {
		//	if (*status == TOEVICT)
		//		s_bid[warpid] = -1;
		//	else
		//		s_bid[warpid] = atomicAdd(cont_subtask + warpid, 1); //subtask_id
		//}
			
		__syncthreads();
		
		//if (s_bid[warpid] >= num_subtask || s_bid[warpid] == -1)
		if (s_bid >=num_subtask || s_bid ==-1) /* If all subtasks have been executed */
			return;
	
		for (int j=0; j<iter_per_subtask; j++) {
			
			 i = s_bid * blockDim.x * iter_per_subtask +  j * blockDim.x + threadIdx.x;
			
			if (i < numelements)
					C[i] = A[i] + B[i];
		}
		
	}

}

__global__ void
preempt_SMT_vectorAdd(const float *A, const float *B, float *C, int numelements, 
			int SIMD_min, int SIMD_max,
			int num_subtask, int iter_per_subtask, int *cont_subtask, State *status)
{
	int i;
	
	__shared__ int s_bid;
	
	unsigned int SM_id = get_smid_VA();
	
	//int warpid = threadIdx.x >> 5;
	
	//int thIdxwarp = threadIdx.x & 0x1F;
	
	if (SM_id <SIMD_min || SM_id > SIMD_max) /* Only blocks executing within SIMD_min and SIMD_max can progress */ 
			return;
			
	#ifdef SHOW_SM
		if (threadIdx.x == 0) 
			printf("%d, VA\n", SM_id);
	#endif
			
	//if (threadIdx.x==0) // Ojo, esto es una prueba. Habría que tener en cuenta iteraciones entre distintos bloques
	//	*status = RUNNING;
		
	while (1){
		
		/********** Task Id calculation *************/
		
		if (threadIdx.x == 0) {
			if (*status == TOEVICT)
				s_bid = -1;
			else
				s_bid = atomicAdd(cont_subtask, 1); //subtask_id
		}
		
		//if ( thIdxwarp == 0) {
		//	if (*status == TOEVICT)
		//		s_bid[warpid] = -1;
		//	else
		//		s_bid[warpid] = atomicAdd(cont_subtask + warpid, 1); //subtask_id
		//}
			
		__syncthreads();
		
		//if (s_bid[warpid] >= num_subtask || s_bid[warpid] == -1)
		if (s_bid >=num_subtask || s_bid ==-1) /* If all subtasks have been executed */
			return;
	
		for (int j=0; j<iter_per_subtask; j++) {
			
			 i = s_bid * blockDim.x * iter_per_subtask +  j * blockDim.x + threadIdx.x;
			
			if (i < numelements)
					C[i] = A[i] + B[i];
		}
		
	}

}

__global__ void
memaddr_preempt_SMT_vectorAdd(const float *A, const float *B, float *C, int numelements, 
			int *numUniqueAddr, int SIMD_min, int SIMD_max,
			int num_subtask, int iter_per_subtask, int *cont_subtask, State *status)
{
	int i;
	
	__shared__ int s_bid;
	
	unsigned int SM_id = get_smid_VA();
	
	//int warpid = threadIdx.x >> 5;
	
	//int thIdxwarp = threadIdx.x & 0x1F;
	
	if (SM_id <SIMD_min || SM_id > SIMD_max) /* Only blocks executing within SIMD_min and SIMD_max can progress */ 
			return;
			
	//if (threadIdx.x==0) // Ojo, esto es una prueba. Habría que tener en cuenta iteraciones entre distintos bloques
	//	*status = RUNNING;
		
	while (1){
		
		/********** Task Id calculation *************/
		
		if (threadIdx.x == 0) {
			if (*status == TOEVICT)
				s_bid = -1;
			else
				s_bid = atomicAdd(cont_subtask, 1); //subtask_id
		}
		
		//if ( thIdxwarp == 0) {
		//	if (*status == TOEVICT)
		//		s_bid[warpid] = -1;
		//	else
		//		s_bid[warpid] = atomicAdd(cont_subtask + warpid, 1); //subtask_id
		//}
			
		__syncthreads();
		
		//if (s_bid[warpid] >= num_subtask || s_bid[warpid] == -1)
		if (s_bid >=num_subtask || s_bid ==-1) /* If all subtasks have been executed */
			return;
	
		for (int j=0; j<iter_per_subtask; j++) {
			
			 i = s_bid * blockDim.x * iter_per_subtask +  j * blockDim.x + threadIdx.x;
			
			if (i < numelements)
			{
#if defined(COUNT_ALL_TASKS)

				if ( s_bid == 0 )
#endif
				{
					get_unique_lines((intptr_t) &A[i], numUniqueAddr);
					get_unique_lines((intptr_t) &B[i], numUniqueAddr);
					get_unique_lines((intptr_t) &C[i], numUniqueAddr);
				}
				C[i] = A[i] + B[i];
			}
		}
		
	}

}

__global__ void
profiling_SMT_vectorAdd(const float *A, const float *B, float *C, int numelements, 						
						int num_subtask,
						int iter_per_subtask,
						int *cont_SM,
						int *cont_subtask,
						State *status)
{
	int i;
	
	__shared__ int s_bid, CTA_cont;
	
	unsigned int SM_id = get_smid_VA();
	
	if (SM_id >= 8){ /* Only blocks executing in first 8 SM  are used for profiling */ 
		//delay();
		return;
	}
	
	if (threadIdx.x == 0) {
		CTA_cont = atomicAdd(&cont_SM[SM_id], 1);
	//	if (SM_id == 7 && CTA_cont == 8)
	//		printf("Aqui\n");
	}
	
	__syncthreads();
	
	if (CTA_cont > SM_id) {/* Only one block makes computation in SM0, two blocks in SM1 and so on */
		//delay();
		return;
	}
	
	//if (threadIdx.x == 0)
	//	printf ("SM=%d CTA = %d\n", SM_id, CTA_cont);

	int cont_task = 0;
		
	while (1){
		
		/********** Task Id calculation *************/
		
		if (threadIdx.x == 0) {
			if (*status == TOEVICT)
				s_bid = -1;
			else
				s_bid = atomicAdd(cont_subtask, 1); //subtask_id
		}
		
		//if ( thIdxwarp == 0) {
		//	if (*status == TOEVICT)
		//		s_bid[warpid] = -1;
		//	else
		//		s_bid[warpid] = atomicAdd(cont_subtask + warpid, 1); //subtask_id
		//}
			
		__syncthreads();
		
		//if (s_bid[warpid] >= num_subtask || s_bid[warpid] == -1)
		if (s_bid >=num_subtask || s_bid ==-1){ /* If all subtasks have been executed */
			if (threadIdx.x == 0)
				printf ("SM=%d CTA=%d Executed_tasks= %d \n", SM_id, CTA_cont, cont_task);
			return;
		}
		
		if (threadIdx.x == 0) // Acumula numeor de tareas ejecutadas
			 cont_task++;
	
		for (int j=0; j<iter_per_subtask; j++) {
			
			 i = s_bid * blockDim.x * iter_per_subtask +  j * blockDim.x + threadIdx.x;
			
			if (i < numelements)
				C[i] = A[i] + B[i];
		}
		
	}

}

/**
 * Host main routine
 */
 
//// Global variables

 // float *h_A;
 // float *h_B;
 // float *h_C;
 // float *d_A;
 // float *d_B;
 // float *d_C;
 
 // int numElements;
 
int VA_start_kernel_dummy(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_VA_params * params = (t_VA_params *)kstub->params;
	
	// Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    /*int*/ params->numElements = kstub->kconf.gridsize.x * kstub->kconf.blocksize.x * kstub->kconf.coarsening;
    size_t size = params->numElements * sizeof(float);
    //printf("[Vector addition of %d elements]\n", params->numElements);

    // Allocate the host input vector A
    params->h_A = (float *)malloc(size);

    // Allocate the host input vector B
    params->h_B = (float *)malloc(size);

    // Allocate the host output vector C
    params->h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (params->h_A == NULL || params->h_B == NULL || params->h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < params->numElements; ++i)
    {
        params->h_A[i] = rand()/(float)RAND_MAX;
        params->h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    params->d_A = NULL;
    err = hipMalloc((void **)&params->d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    params->d_B = NULL;
    err = hipMalloc((void **)&params->d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    params->d_C = NULL;
    err = hipMalloc((void **)&params->d_C, size);
    checkCudaErrors(hipMemset(params->d_C, 0, size));


    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    //printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(params->d_A, params->h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(params->d_B, params->h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
	return 0;
}

int VA_start_mallocs(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_VA_params * params = (t_VA_params *)kstub->params;
	
	// Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    /*int*/ params->numElements = kstub->kconf.gridsize.x * kstub->kconf.blocksize.x * kstub->kconf.coarsening;
    size_t size = params->numElements * sizeof(float);
    //printf("[Vector addition of %d elements]\n", params->numElements);
	
#if defined(MEMCPY_SYNC) || defined(MEMCPY_ASYNC)

    // Allocate the host input vector A
	hipHostMalloc(&params->h_A, size);

    // Allocate the host input vector B
    hipHostMalloc(&params->h_B, size);

    // Allocate the host output vector C
    hipHostMalloc(&params->h_C, size);
	
	// Allocate the device input vector A
    params->d_A = NULL;
    err = hipMalloc((void **)&params->d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    params->d_B = NULL;
    err = hipMalloc((void **)&params->d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    params->d_C = NULL;
    err = hipMalloc((void **)&params->d_C, size);
    checkCudaErrors(hipMemset(params->d_C, 0, size));


    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
#else
	#ifdef MANAGED_MEM

	hipMallocManaged(&params->h_A, size);
	hipMallocManaged(&params->h_B, size);
	hipMallocManaged(&params->h_C, size);
	
	params->d_A = params->h_A;
	params->d_B = params->h_B;
	params->d_C = params->h_C;
	
	#else
		printf("No transfer model: Exiting ...\n");
		exit(-1);
	#endif
#endif


    // Verify that allocations succeeded
    if (params->h_A == NULL || params->h_B == NULL || params->h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < params->numElements; ++i)
    {
        params->h_A[i] = rand()/(float)RAND_MAX;
        params->h_B[i] = rand()/(float)RAND_MAX;
		params->h_C[i] = 0;
    }

	return 0;
}
	
int VA_start_transfers(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_VA_params * params = (t_VA_params *)kstub->params;

	size_t size = params->numElements * sizeof(float);
	
	// Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
	
#ifdef MEMCPY_SYNC
	// Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    //printf("Copy input data from the host memory to the CUDA device\n");
  /*  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }*/
	
	/*HtD_data_transfer(d_A, h_A, size, C_S);*/
	
	enqueue_tcomamnd(tqueues, params->d_A, params->h_A, size, hipMemcpyHostToDevice, 0, BLOCKING, DATA, LOW, kstub);

    /*err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }*/
	
	//HtD_data_transfer(d_B, h_B, size, C_S);
	enqueue_tcomamnd(tqueues, params->d_B, params->h_B, size, hipMemcpyHostToDevice, 0, BLOCKING, DATA, LOW, kstub);
	
	kstub->HtD_tranfers_finished = 1;

	
#else
	
	#ifdef MEMCPY_ASYNC
	
	err = hipMemcpyAsync(params->d_A, params->h_A, size, hipMemcpyHostToDevice, kstub->transfer_s[0]);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpyAsync(params->d_B, params->h_B, size, hipMemcpyHostToDevice, kstub->transfer_s[0]);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
	//hipEventRecord(kstub->end_HtD, kstub->transfer_s[0]);
	hipStreamSynchronize(kstub->transfer_s[0]);
	
	/*
	enqueue_tcomamnd(tqueues, d_A, h_A, size, hipMemcpyHostToDevice, kstub->transfer_s[0], NONBLOCKING, DATA, MEDIUM, kstub);
	enqueue_tcomamnd(tqueues, d_B,  h_B, size, hipMemcpyHostToDevice, kstub->transfer_s[0], NONBLOCKING, LAST_TRANSFER, MEDIUM, kstub);
*/
	//enqueue_tcomamnd(tqueues, NULL, NULL, 0, hipMemcpyHostToDevice, kstub->transfer_s[0], STREAM_SYNCHRO, DATA, MEDIUM, kstub);
	
	
	//kstub->HtD_tranfers_finished = 1;

	#else
	#ifdef MANAGED_MEM

	hipDeviceProp_t p;
    hipGetDeviceProperties(&p, kstub->deviceId);
	
	if (p.concurrentManagedAccess)
	{
		err = hipMemPrefetchAsync(params->h_A, size, kstub->deviceId);
		if ( err != hipSuccess) {
			printf("Error in vAdd:hipMemPrefetchAsync\n");
			exit(EXIT_FAILURE);
		}
		err = hipMemPrefetchAsync(params->h_B, size, kstub->deviceId);
		if ( err != hipSuccess) {
			printf("Error in vAdd:hipMemPrefetchAsync\n");
			exit(EXIT_FAILURE);
		}
		err = hipMemPrefetchAsync(params->h_C, size, kstub->deviceId);
		if ( err != hipSuccess) {
			printf("Error in vAdd:hipMemPrefetchAsync\n");
			exit(EXIT_FAILURE);
		}
	}
	
	//hipEventRecord(kstub->end_HtD, kstub->transfer_s[0]);
	
	//hipStreamSynchronize(kstub->transfer_s[0]);
	kstub->HtD_tranfers_finished = 1;

	#endif
	#endif
#endif

	return 0;
}
	
 
int VA_end_kernel_dummy(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_VA_params * params = (t_VA_params *)kstub->params;
	
#ifdef MEMCPY_SYNC

	hipEventSynchronize(kstub->end_Exec);

	// Copy the device result vector in device memory to the host result vector
    // in host memory.
    //printf("Copy output data from the CUDA device to the host memory\n");
    
	/* hipError_t err = hipSuccess;
	 err = hipMemcpy(h_C, d_C, numElements*sizeof(float), hipMemcpyDeviceToHost);

     if (err != hipSuccess)
     {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
     }*/
	 
	 //DtH_data_transfer(h_C, d_C, numElements*sizeof(float), C_S);
	 enqueue_tcomamnd(tqueues, params->h_C, params->d_C, params->numElements*sizeof(float), hipMemcpyDeviceToHost, 0, BLOCKING, DATA, LOW, kstub);
	 
#else
	#ifdef MEMCPY_ASYNC

	hipError_t err = hipSuccess;

	//err = hipEventSynchronize(kstub->end_Exec);
	
	err = hipMemcpyAsync(params->h_C, params->d_C, params->numElements*sizeof(float), hipMemcpyDeviceToHost, kstub->transfer_s[1]);

     if (err != hipSuccess)
     {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
     }
	 			printf("-->Comienzo de DtH para tarea %d\n", kstub->id);

	//enqueue_tcomamnd(tqueues, h_C, d_C, numElements*sizeof(float), hipMemcpyDeviceToHost, kstub->transfer_s[1] , NONBLOCKING, LAST_TRANSFER, MEDIUM, kstub);
	//enqueue_tcomamnd(tqueues, NULL, NULL, 0, hipMemcpyDeviceToHost, kstub->transfer_s[1], STREAM_SYNCHRO, DATA, MEDIUM, kstub);
	 
	//kstub->DtH_tranfers_finished = 1;
	
	//printf("-->Fin de DtH para tarea %d\n", kstub->id);

	 //hipEventRecord(kstub->end_DtH, kstub->transfer_s[1]);
	
	/*hipEventSynchronize(kstub->end_DtH);*/
	
	#else
		#ifdef MANAGED_MEM
			hipStreamSynchronize(*(kstub->execution_s)); // To be sure kernel execution has finished before processing output data
		#endif
	#endif
#endif

/*
    // // Verify that the result vector is correct
     for (int i = 0; i < numElements; ++i)
     {
         if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
         {
             fprintf(stderr, "Result verification failed at element %d!\n", i);
             exit(EXIT_FAILURE);
         }
     }*/
    //printf("Test PASSED\n");
	
	/*
	
#if defined(MEMCPY_SYNC) || defined(MEMCPY_ASYNC)

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    // Reset the device and exit
    //err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
#else
	
	hipFree(h_A);
	hipFree(h_B);
	hipFree(h_C);
#endif
	*/
	return 0;

}	 



int launch_orig_VA(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_VA_params * params = (t_VA_params *)kstub->params;
	
	original_vectorAdd<<<kstub->kconf.gridsize.x, kstub->kconf.blocksize.x, 0, *(kstub->execution_s)>>>(
		params->d_A, params->d_B, params->d_C, 
		kstub->kconf.coarsening,
		params->numElements);
	
	return 0;
}

int prof_VA(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_VA_params * params = (t_VA_params *)kstub->params;
	
	profiling_SMT_vectorAdd<<<kstub->kconf.numSMs * kstub->kconf.max_persistent_blocks, kstub->kconf.blocksize.x, 0, *(kstub->execution_s)>>>/*(VAparams->d_A, VAparams->d_B, VAparams->d_C, VAparams->numElements, */
			(params->d_A, params->d_B, params->d_C, params->numElements,
			kstub->total_tasks,
			kstub->kconf.coarsening,
			kstub->d_SMs_cont,
			kstub->d_executed_tasks,
			&kstub->gm_state[kstub->stream_index]);
	return 0;
}

int launch_preemp_VA(void *arg)
{
	t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_VA_params * params = (t_VA_params *)kstub->params;
	
	#ifdef SMT

	if ( !(kstub->memaddr_profile) )
		preempt_SMT_vectorAdd<<<kstub->kconf.numSMs * kstub->kconf.max_persistent_blocks, kstub->kconf.blocksize.x, 0, *(kstub->execution_s)>>>/*(VAparams->d_A, VAparams->d_B, VAparams->d_C, VAparams->numElements, */
			(params->d_A, params->d_B, params->d_C, params->numElements,
			kstub->idSMs[0],
			kstub->idSMs[1],
			kstub->total_tasks,
			kstub->kconf.coarsening,
			kstub->d_executed_tasks,
			&(kstub->gm_state[kstub->stream_index]));
	else
		memaddr_preempt_SMT_vectorAdd<<<kstub->kconf.numSMs * kstub->kconf.max_persistent_blocks, kstub->kconf.blocksize.x, 0, *(kstub->execution_s)>>>/*(VAparams->d_A, VAparams->d_B, VAparams->d_C, VAparams->numElements, */
			(params->d_A, params->d_B, params->d_C, params->numElements,
			kstub->d_numUniqueAddr,
			kstub->idSMs[0],
			kstub->idSMs[1],
			kstub->total_tasks,
			kstub->kconf.coarsening,
			kstub->d_executed_tasks,
			&(kstub->gm_state[kstub->stream_index]));
	
	#else
		
	preempt_SMK_vectorAdd<<<kstub->kconf.numSMs * kstub->kconf.max_persistent_blocks, kstub->kconf.blocksize.x, 0, *(kstub->execution_s)>>>
			(params->d_A, params->d_B, params->d_C, params->numElements, 
			kstub->num_blocks_per_SM,
			kstub->total_tasks,
			kstub->kconf.coarsening,
			kstub->d_SMs_cont,
			kstub->d_executed_tasks,
			&(kstub->gm_state[kstub->stream_index])
	);	
		
	#endif
	
	return 0;
}