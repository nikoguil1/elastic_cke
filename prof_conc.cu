#include <unistd.h>
#include <time.h>
#include <string.h>
#include <unistd.h>
#include <helper_functions.h>   // helper functions for string parsing
#include <hip/hip_runtime_api.h>   
#include "elastic_kernel.h"

int main(int argc, char **argv)
{

	hipError_t err;
	int deviceId = atoi(argv[1]);
	// Select device
	hipSetDevice(deviceId);
	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);	
	printf("Device=%s\n", deviceProp.name);
	
	/** Create commom streams for all kernels: two for asynchronous transfers, one for preemption commands*/
	hipStream_t *transfers_s;
	transfers_s = (hipStream_t *)calloc(2, sizeof(hipStream_t));
	
	for (int i=0;i<2;i++){
		err = hipStreamCreate(&transfers_s[i]);
		checkCudaErrors(err);
	} 
	
	hipStream_t preemp_s;
	checkCudaErrors(hipStreamCreateWithFlags(&preemp_s, hipStreamNonBlocking)); 
	
	// Create kerbel stub
	t_kernel_stub *kstub;
	
	create_stubinfo(&kstub, deviceId, BS, transfers_s, &preemp_s);
	
	// Make transfoer
	(kstub->startMallocs)((void *)(kstub));
	(kstub->startTransfers)((void *)(kstub));
	hipDeviceSynchronize();
	
	// Execute 
	
	int idSMs[2];
	idSMs[0]=0;idSMs[1]=kstub->kconf.numSMs-1;
	kstub->idSMs = idSMs;	
	//(kstub->launchCKEkernel)(kstub);
	prof_BS(kstub);
	
	hipDeviceSynchronize();

	return 0;
}
	
	
	
	
	
	