#include "hip/hip_runtime.h"
#include <stdio.h>          /* printf()                 */
#include <stdlib.h>         /* exit(), malloc(), free() */
#include <unistd.h>
#include <sys/wait.h>
#include <sys/types.h>      /* key_t, sem_t, pid_t      */
#include <sys/shm.h>        /* shmat(), IPC_RMID        */
#include <sys/mman.h>		/* mmap						*/
#include <errno.h>          /* errno, ECHILD            */
#include <semaphore.h>      /* sem_open(), sem_destroy(), sem_wait().. */
#include <fcntl.h>          /* O_CREAT, O_EXEC          */
#include <pthread.h>
#include <time.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>   // helper functions for string parsing
#include <hip/hip_runtime_api.h>   
#include "elastic_kernel.h"

int change_thread_percentage(int percentage) {
	FILE *server_list = NULL;
	char server_string[256], command_string[256];
	int server_pid;
	server_list = popen("echo get_server_list | nvidia-cuda-mps-control", "r");
	if (!server_list)
	{
		perror("Error reading MPS server list");
		exit(-1);
	}
	fgets(server_string, 1000, server_list);
	while (!feof(server_list))
	{
		server_pid = atoi(server_string);
		fgets(server_string, 1000, server_list);
	}
	sprintf(command_string, "echo set_active_thread_percentage %d %d | nvidia-cuda-mps-control > /dev/null", server_pid, percentage);
	//printf("%s\n", command_string);
	int status = system(command_string);
	return(status);
}

int run_original(t_kernel_stub *kstub, double *exectime_s)
{
	//hipEvent_t start, stop;
	//float elapsedTime;
	
	//hipEventCreate(&start);
	//hipEventRecord(start, 0);
	
	kstub->launchORIkernel(kstub);
	hipDeviceSynchronize();
	
	//hipEventCreate(&stop);
	//hipEventRecord(stop,0);
	//hipEventSynchronize(stop);

	//hipEventElapsedTime(&elapsedTime, start, stop);
	
	//*exectime_s = (double)elapsedTime/1000;
	
	return 0;
}


typedef struct{
	t_kernel_stub **kstubs;
	int index; // Index in kstubs array
}t_args;

void *launch_app(void *arg)
{
	t_args *args;
	
	args = (t_args *)arg;
	int index = args->index;
	t_kernel_stub *kstub = args->kstubs[index];
	
	printf("Launching kid=%d\n", kstub->id);
	
	int deviceId = 2;
	hipSetDevice(deviceId);
	
	double exec_time;
	run_original(kstub, &exec_time);
	
	 if (kstub->id == RCONV) {
		kstub = args->kstubs[index + 1];
		run_original(kstub, &exec_time);
	 }
	 
	 if (kstub->id == GCEDD) {
		kstub = args->kstubs[index + 1];
		run_original(kstub, &exec_time);
		
		kstub = args->kstubs[index + 2];
		run_original(kstub, &exec_time);
		
		kstub = args->kstubs[index + 3];
		run_original(kstub, &exec_time);
	 }
	 
	 pthread_exit(NULL);
}

int hyperQ_threads()
{
	
	t_Kernel kid[9];
	kid[0]=MM;
	kid[1]=VA;
	kid[2]=BS;
	kid[3]=Reduction;
	kid[4]=PF;
	kid[5]=GCEDD; // Ojo: en profiling.cu se procesan tambien los tres kernels restantes de la aplicacion
	kid[6]=SPMV_CSRscalar;
	kid[7]=RCONV; // Ojo: en profiling se procesa tambien CCONV
	kid[8]=HST256;
	
	int num_kernels = 2;
	
	// context and streams
	
	hipError_t err;

	// Select device
	int deviceId = 2;
	hipSetDevice(deviceId);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, deviceId);	
	printf("Device=%s\n", deviceProp.name);
	
	/** Create commom streams for all kernels: two for asynchronous transfers, one for preemption commands*/
	hipStream_t *transfers_s;
	transfers_s = (hipStream_t *)calloc(2, sizeof(hipStream_t));
	
	for (int i=0;i<2;i++){
		err = hipStreamCreate(&transfers_s[i]);
		checkCudaErrors(err);
	}
	
	hipStream_t preemp_s;
	checkCudaErrors(hipStreamCreateWithFlags(&preemp_s, hipStreamNonBlocking)); 
		
	// Create kstbus
	int cont = 0;			
	t_kernel_stub **kstubs = (t_kernel_stub **)calloc(13, sizeof(t_kernel_stub*)); // 13 is the max number of kernels for all app
	
	int index[9];
	for (int i=0; i< num_kernels; i++) {
		
		index[i] = cont;
		create_stubinfo(&kstubs[cont], deviceId, kid[i], transfers_s, &preemp_s);
		cont++;
		
		if (kid[i] == RCONV){ // RCONV params struct must be passed to CCONV 
			create_stubinfo_with_params(&kstubs[cont], deviceId, CCONV, transfers_s, &preemp_s, (void *)kstubs[cont-1]->params);
			cont++;
		}
		
		if (kid[i] == GCEDD){
			create_stubinfo_with_params(&kstubs[cont], deviceId, SCEDD, transfers_s, &preemp_s, (void *)kstubs[cont-1]->params);
			cont++;
			
			create_stubinfo_with_params(&kstubs[cont], deviceId, NCEDD, transfers_s, &preemp_s, (void *)kstubs[cont-2]->params);
			cont++;
			
			create_stubinfo_with_params(&kstubs[cont], deviceId, HCEDD, transfers_s, &preemp_s, (void *)kstubs[cont-3]->params);
			cont++;
		}
	}
		
	// make HtD transfers of all kernels
	make_transfers(kstubs, cont);
	
	// Create threads to lauch app
	
	t_args args[9];
	for (int i=0; i<9; i++)
		args[i].kstubs = kstubs;
	pthread_t *thid = (pthread_t *) calloc(num_kernels, sizeof(pthread_t));
	for (int i=0; i<num_kernels; i++) {
		args[i].index = index[i];	
		pthread_create(&thid[i], NULL, launch_app, &args[i]);
	}
	
	for (int i=0; i<num_kernels; i++)
		pthread_join(thid[i], NULL);
	
	hipDeviceSynchronize();
	
	return 0;
}
		

int main (int argc, char **argv)
{
	
	int it;                        /*      loop variables          */
    key_t shmkey;                 /*      shared memory key       */
    int shmid;                    /*      shared memory id        */
    sem_t *sem;                   /*      synch semaphore         *//*shared */
    pid_t pid;                    /*      fork pid                */
	pid_t childs_pid[2];
    int *p;                       /*      shared variable         *//*shared */
    unsigned int value;           /*      semaphore value         */
	
	double ProfilingTimeThreshold = 10.0; // Kernels are launched many times during this interval
//	hyperQ_threads();
//	return 0;


    /* initialize a shared variable in shared memory */
    shmkey = ftok ("/dev/null", 5);       /* valid directory name and a number */
    //printf ("shmkey for p = %d\n", shmkey);
    shmid = shmget (shmkey, sizeof (int), 0644 | IPC_CREAT);
    if (shmid < 0){                           /* shared memory error check */
        perror ("shmget\n");
        exit (1);
    }

    p = (int *) shmat (shmid, NULL, 0);   /* attach p to shared memory */
    *p = 0;
    //printf ("p=%d is allocated in shared memory.\n\n", *p);

    /********************************************************/

    /* initialize semaphores for shared processes */
    sem = sem_open ("pSem", O_CREAT | O_EXCL, 0644, 1); // Binary semaphore 
    /* name of semaphore is "pSem", semaphore is reached using this name */

    //printf ("semaphores initialized.\n\n");
	
	/*// kstubs
	
	hipError_t err;

	// Select device
	int deviceId = 2;
	hipSetDevice(deviceId);
	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);	
	printf("Device=%s\n", deviceProp.name);
	
	// Create commom streams for all kernels: two for asynchronous transfers, one for preemption commands
	hipStream_t *transfers_s;
	transfers_s = (hipStream_t *)calloc(2, sizeof(hipStream_t));
	
	for (int i=0;i<2;i++){
		err = hipStreamCreate(&transfers_s[i]);
		checkCudaErrors(err);
	} 
	
	hipStream_t preemp_s;
	checkCudaErrors(hipStreamCreateWithFlags(&preemp_s, hipStreamNonBlocking)); */

	t_Kernel kid[9];
	int index[9];
	kid[0]=MM;
	kid[1]=VA;
	kid[2]=BS;
	kid[3]=Reduction;
	kid[4]=PF;
	kid[5]=GCEDD; // Ojo: en profiling.cu se procesan tambien los tres kernels restantes de la aplicacion
	kid[6]=SPMV_CSRscalar;
	kid[7]=RCONV; // Ojo: en profiling se procesa tambien CCONV
	kid[8]=HST256;
	
	if ( argc > 2 ) kid[0] = kid_from_name(argv[2]);
	if ( argc > 3 )	kid[1] = kid_from_name(argv[3]);

	int num_kernels = 2;

	int percentage = 50;
	if ( argc > 4 ) percentage = atoi(argv[4]);
	change_thread_percentage(percentage);

	/*for (int i=0; i<num_kernels; i++){
		total_num_kernels++;
		if (kid[i] == RCONV) total_num_kernels++;
		if (kid[i] == GCEDD) total_num_kernels += 3;
	}*/
	
	/** Create stubs ***/
	// Ojo la lista de kernels sólo debe ponerse el primero de una aplicacion. Los demás
	// son creados por el siguiente código
	/*t_kernel_stub **kstubs = (t_kernel_stub **)calloc(total_num_kernels, sizeof(t_kernel_stub*));
	for (int i=0, cont=0; i<num_kernels; i++) {	
		create_stubinfo(&kstubs[cont], deviceId, kid[i], transfers_s, &preemp_s);
		index[i] = cont;
		cont++;
		if (kid[i] == RCONV){ // RCONV params struct must be passed to CCONV 
			create_stubinfo_with_params(&kstubs[cont], deviceId, CCONV, transfers_s, &preemp_s, (void *)kstubs[cont-1]->params);
			cont++;
		}
		
		if (kid[i] == GCEDD){
			create_stubinfo_with_params(&kstubs[cont], deviceId, SCEDD, transfers_s, &preemp_s, (void *)kstubs[cont-1]->params);
			cont++;
			
			create_stubinfo_with_params(&kstubs[cont], deviceId, NCEDD, transfers_s, &preemp_s, (void *)kstubs[cont-2]->params);
			cont++;
			
			create_stubinfo_with_params(&kstubs[cont], deviceId, HCEDD, transfers_s, &preemp_s, (void *)kstubs[cont-3]->params);
			cont++;
		}
	}

	// make HtD transfers of all kernels
	make_transfers(kstubs, total_num_kernels);*/
   
	void *message1 = mmap(NULL, 1024, PROT_READ | PROT_WRITE, MAP_SHARED | MAP_ANONYMOUS, -1, 0);
	void *message2 = mmap(NULL, 1024, PROT_READ | PROT_WRITE, MAP_SHARED | MAP_ANONYMOUS, -1, 0);

    /* fork child processes */
    for (it = 0; it < num_kernels; it++){
        pid = fork ();
        if (pid < 0) {
        /* check for error      */
            sem_unlink ("pSem");   
            sem_close(sem);  
            /* unlink prevents the semaphore existing forever */
            /* if a crash occurs during the execution         */
            printf ("Fork error.\n");
        }
        else if (pid == 0)
            break;                  /* child processes */
		childs_pid[it] = pid;
		char kname[100];
		kid_from_index(kid[it], kname);
		//printf("Child %d with PID %d will launch %s\n", it, childs_pid[it], kname);
    }

    /******************************************************/
    /******************   PARENT PROCESS   ****************/
    /******************************************************/
    if (pid != 0){
		unsigned long int numLaunchs[2], tmp0, tmp1, tmp2;
		float mps_time[2];
		float tmp3;
/*
		close( descr[1] ); // Close output descriptor
		read( descr[0], message1, 1000);
		sscanf(message1, "%d %d %lu %f", &tmp0, &tmp1, &tmp2, &tmp3);
		printf("Message1: %s <-> %d %d %lu %f\n", message1, tmp0, tmp1, tmp2, tmp3);
		if ( tmp0 == childs_pid[0] ) {		
			numLaunchs[0] = tmp2;
			mps_time[0] = tmp3;
		}
		else if ( tmp0 == childs_pid[1] ) {
			numLaunchs[1] = tmp2;
			mps_time[1] = tmp3;
		}
		read( descr[0], message2, 1000);
		sscanf(message2, "%d %d %lu %f", &tmp0, &tmp1, &tmp2, &tmp3);//		printf("Message2: %s <-> %d %d %f\n", message2, tmp1, tmp2, tmp3);
		printf("Message2: %s <-> %d %d %lu %f\n", message2, tmp0, tmp1, tmp2, tmp3);
		if ( tmp0 == childs_pid[0] ) {		
			numLaunchs[0] = tmp2;
			mps_time[0] = tmp3;
		}
		else if ( tmp0 == childs_pid[1] ) {
			numLaunchs[1] = tmp2;
			mps_time[1] = tmp3;
		}
		close( descr[0] );
*/
        /* wait for all children to exit */
		int retval;
        while (pid = waitpid (-1, &retval, 0)){
//			printf ("\n%d returns %d\n", pid, WEXITSTATUS(retval));
/*			if ( childs_pid[0] == pid )
				numLaunchs[0] = WEXITSTATUS(retval);
			else if ( childs_pid[1] == pid )
				numLaunchs[1] = WEXITSTATUS(retval);
*/
            if (errno == ECHILD)
                break;
        }

        //printf ("\nParent: All children have exited\n");
		sscanf((char *)message1, "%lu %lu %lu %f", &tmp0, &tmp1, &tmp2, &tmp3);
//		printf("Message1: %s <-> %lu %lu %lu %f\n", (char *)message1, tmp0, tmp1, tmp2, tmp3);
		if ( tmp0 == childs_pid[0] ) {		
			numLaunchs[0] = tmp2;
			mps_time[0] = tmp3;
		}
		else if ( tmp0 == childs_pid[1] ) {
			numLaunchs[1] = tmp2;
			mps_time[1] = tmp3;
		}

		sscanf((char *)message2, "%lu %lu %lu %f", &tmp0, &tmp1, &tmp2, &tmp3);//		printf("Message2: %s <-> %d %d %f\n", message2, tmp1, tmp2, tmp3);
//		printf("Message2: %s <-> %lu %lu %lu %f\n", (char *)message2, tmp0, tmp1, tmp2, tmp3);
		if ( tmp0 == childs_pid[0] ) {		
			numLaunchs[0] = tmp2;
			mps_time[0] = tmp3;
		}
		else if ( tmp0 == childs_pid[1] ) {
			numLaunchs[1] = tmp2;
			mps_time[1] = tmp3;
		}

        /* shared memory detach */
        shmdt (p);
        shmctl (shmid, IPC_RMID, 0);

        /* cleanup semaphores */
        sem_unlink ("pSem");   
        sem_close(sem);  
        /* unlink prevents the semaphore existing forever */
        /* if a crash occurs during the execution         */

		change_thread_percentage(100);

		// Select device
		hipError_t err;
		int deviceId = 0;
		if ( argc > 1 ) deviceId = atoi(argv[1]);
		hipSetDevice(deviceId);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, deviceId);	
//		printf("Parent working on Device=%s\n", deviceProp.name);
	
		/** Create commom streams for all kernels: two for asynchronous transfers, one for preemption commands*/
		hipStream_t *transfers_s;
		transfers_s = (hipStream_t *)calloc(2, sizeof(hipStream_t));
	
		for (int i=0;i<2;i++){
			err = hipStreamCreate(&transfers_s[i]);
			checkCudaErrors(err);
		}
	
		hipStream_t preemp_s;
		checkCudaErrors(hipStreamCreateWithFlags(&preemp_s, hipStreamNonBlocking)); 
		
		double max_mps_time = 0, seq_time = 0;
		for ( it = 0; it < num_kernels; it++) {
			char kname[100];
			kid_from_index(kid[it], kname);	
			//printf("Parent: creating kstubs for kernel %s\n", kname);
	
			// Create kstbus
			int cont = 0;
			t_kernel_stub **kstubs = (t_kernel_stub **)calloc(4, sizeof(t_kernel_stub*)); // Four is the man number of kernels of a app
			int status = create_stubinfo(&kstubs[cont], deviceId, kid[it], transfers_s, &preemp_s);
			if ( status < 0 ) {
				printf("Exiting, no stubs created");
				exit(-1);
			}
			cont++;
		
			if (kid[it] == RCONV) { // RCONV params struct must be passed to CCONV 
				create_stubinfo_with_params(&kstubs[cont], deviceId, CCONV, transfers_s, &preemp_s, (void *)kstubs[cont-1]->params);
				cont++;
			}		
			if (kid[it] == GCEDD) {
				create_stubinfo_with_params(&kstubs[cont], deviceId, SCEDD, transfers_s, &preemp_s, (void *)kstubs[cont-1]->params);
				cont++;
				create_stubinfo_with_params(&kstubs[cont], deviceId, NCEDD, transfers_s, &preemp_s, (void *)kstubs[cont-2]->params);
				cont++;			
				create_stubinfo_with_params(&kstubs[cont], deviceId, HCEDD, transfers_s, &preemp_s, (void *)kstubs[cont-3]->params);
				cont++;
			}
			
			// make HtD transfers of all kernels
			//printf("Parent: transfering data\n");
			make_transfers(kstubs, cont);
		
			// Solo original profiling
			//printf("Parent: Launching %lu times %s\n", numLaunchs[it], kname);
			double exectime_s[4];
			struct timespec now;
			double exec_time = 0.0;
			for (int n = 0; n < numLaunchs[it]; n++) {
				clock_gettime(CLOCK_MONOTONIC, &now);
				double time1 = (double)now.tv_sec+(double)now.tv_nsec*1e-9;
				for (int i=0; i < cont; i++) {	
					run_original(kstubs[i], &exectime_s[i]);
				}
				clock_gettime(CLOCK_MONOTONIC, &now);
				double time2 = (double)now.tv_sec+(double)now.tv_nsec*1e-9;	
				exec_time += time2 - time1;
			}
			
			printf("%d\t%s\t%lu\t%f\t%f", percentage, kname, numLaunchs[it], mps_time[it], exec_time);
			if ( mps_time[it] > max_mps_time )
				max_mps_time = mps_time[it];
			seq_time += exec_time;
			if ( it == 1 )
				printf("\t%f\n", seq_time/max_mps_time );
			else
				printf("\n");
		}


        exit (0);
    }

    /******************************************************/
    /******************   CHILD PROCESS   *****************/
    /******************************************************/
    else{
		
		// context and streams
	
		hipError_t err;

		// Select device
		int deviceId = 0;
		if ( argc > 1 ) deviceId = atoi(argv[1]);
		hipSetDevice(deviceId);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, deviceId);	
//		printf("Chidl %d working on Device=%s\n", it, deviceProp.name);
	
		/** Create commom streams for all kernels: two for asynchronous transfers, one for preemption commands*/
		hipStream_t *transfers_s;
		transfers_s = (hipStream_t *)calloc(2, sizeof(hipStream_t));
	
		for (int i=0;i<2;i++){
			err = hipStreamCreate(&transfers_s[i]);
			checkCudaErrors(err);
		}
	
		hipStream_t preemp_s;
		checkCudaErrors(hipStreamCreateWithFlags(&preemp_s, hipStreamNonBlocking)); 
		
//		printf("Child %d creating kstubs for kernel %d\n", it, kid[it]);
	
		// Create kstbus
		int cont = 0;
		t_kernel_stub **kstubs = (t_kernel_stub **)calloc(4, sizeof(t_kernel_stub*)); // Four is the man number of kernels of a app
		int status = create_stubinfo(&kstubs[cont], deviceId, kid[it], transfers_s, &preemp_s);
		if ( status < 0 ) {
			printf("Exiting, no stubs created");
			exit(-1);
		}
		cont++;
		
		if (kid[it] == RCONV){ // RCONV params struct must be passed to CCONV 
			create_stubinfo_with_params(&kstubs[cont], deviceId, CCONV, transfers_s, &preemp_s, (void *)kstubs[cont-1]->params);
			cont++;
		}
		
		if (kid[it] == GCEDD){
			create_stubinfo_with_params(&kstubs[cont], deviceId, SCEDD, transfers_s, &preemp_s, (void *)kstubs[cont-1]->params);
			cont++;
			
			create_stubinfo_with_params(&kstubs[cont], deviceId, NCEDD, transfers_s, &preemp_s, (void *)kstubs[cont-2]->params);
			cont++;
			
			create_stubinfo_with_params(&kstubs[cont], deviceId, HCEDD, transfers_s, &preemp_s, (void *)kstubs[cont-3]->params);
			cont++;
		}
		
		// make HtD transfers of all kernels
		make_transfers(kstubs, cont);
		
//		printf("Child=%d Transferencia terminada\n", it);

		
		// Barrier
   /*     sem_wait (sem);           // P operation 
        printf ("  Child(%d) is in critical section.\n", it);
        //sleep (1);
        *p += 1 ;              //increment *p by 0, 1 or 2 based on i 
        printf ("  Child(%d) new value of *p=%d.\n", it, *p);
        sem_post (sem);           /// V operation 
     */   
		*p += 1;
		while (*p < num_kernels); // Spin lock

		
		// Solo original profiling
	
		char kname[100];
		kid_from_index(kstubs[0]->id, kname);
//		printf("Child %d launches %s (kid %d) \n", it, kname, kstubs[0]->id);
		double exectime_s[4];
		struct timespec now;
		clock_gettime(CLOCK_REALTIME, &now);
		double time0 = (double)now.tv_sec+(double)now.tv_nsec*1e-9;
		double elapsed_time = 0.0, exec_time = 0.0;
		unsigned long int numLaunchs = 0;
		while ( elapsed_time < ProfilingTimeThreshold ) {
			for (int i=0; i < cont; i++) {	
				clock_gettime(CLOCK_REALTIME, &now);
				double time1 = (double)now.tv_sec+(double)now.tv_nsec*1e-9;
				run_original(kstubs[i], &exectime_s[i]);
				clock_gettime(CLOCK_REALTIME, &now);
				double time2 = (double)now.tv_sec+(double)now.tv_nsec*1e-9;	
				exec_time += time2 - time1;
				elapsed_time = time2 - time0;
			}
			numLaunchs++;
		}


//		printf("Child %d ha lanzado %d veces el kernel %s : exectime=%f\n", it, numLaunchs, kname, exec_time);
		//printf("\t\t\tChild: %s launched %d times, exectime=%f\n", kname, numLaunchs, exec_time);
		char *child_message;
		if ( it == 0 ) child_message = (char *) message1;
		else  child_message = (char *) message2;
		sprintf(child_message, "%lu %lu %lu %f", (ulong) getpid(), (ulong) kid[it], (ulong) numLaunchs, exec_time);
//		sprintf((char *) message1, "%d %d %lu %f\n", getpid(), kid[it], numLaunchs, exec_time);
//		printf("%s <<>> %d %d %lu %f\n", child_message, getpid(), kid[it], numLaunchs, exec_time);
		/*
		if (kid[it] == GCEDD) {
			double exectime_s[4];
			for (int i=0; i < 4; i++) 
				run_original(kstubs[i], &exectime_s[i]);
		}
		else if (kid[index[it]] == RCONV) {
			double exectime_s[2];
			for (int i=0; i < 2; i++) 
				run_original(kstubs[i], &exectime_s[i]);
		}
		else {
			double exectime_s;
			run_original(kstubs[0], &exectime_s);
			printf("Child %d lanzando kernel %d. Tiempo=%f\n", it,  kid[it], exectime_s);
		}
		*/
		hipDeviceSynchronize();
/*
		close( descr[0] ); // Close output descriptor
		write( descr[1], child_message, strlen(child_message));
		close( descr[1] );
*/
		exit(numLaunchs);
    }
}